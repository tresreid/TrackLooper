#include "hip/hip_runtime.h"
# include "Segment.cuh"
//#ifdef CACHE_ALLOC
#include "allocate.h"
//#endif

///FIXME:NOTICE THE NEW maxPixelSegments!

void SDL::createSegmentsInUnifiedMemory(struct segments& segmentsInGPU, unsigned int maxSegments, unsigned int nModules, unsigned int maxPixelSegments)
{
    //FIXME:Since the number of pixel segments is 10x the number of regular segments per module, we need to provide
    //extra memory to the pixel segments
    unsigned int nMemoryLocations = maxSegments * (nModules - 1) + maxPixelSegments;
#ifdef CACHE_ALLOC
    hipStream_t stream=0; 
    segmentsInGPU.mdIndices = (unsigned int*)cms::cuda::allocate_managed(nMemoryLocations*6 *sizeof(unsigned int),stream);
    segmentsInGPU.nSegments = (unsigned int*)cms::cuda::allocate_managed(nModules *sizeof(unsigned int),stream);
    segmentsInGPU.dPhis = (float*)cms::cuda::allocate_managed((nMemoryLocations*6 + maxPixelSegments * 8) *sizeof(float),stream);
    segmentsInGPU.superbin = (int*)cms::cuda::allocate_managed((maxPixelSegments) *sizeof(int),stream);
    segmentsInGPU.pixelType = (int*)cms::cuda::allocate_managed((maxPixelSegments) *sizeof(int),stream);
#else
    hipMallocManaged(&segmentsInGPU.mdIndices, nMemoryLocations * 6 * sizeof(unsigned int));
    hipMallocManaged(&segmentsInGPU.nSegments, nModules * sizeof(unsigned int));
    hipMallocManaged(&segmentsInGPU.dPhis, (nMemoryLocations * 6 + maxPixelSegments * 8)*sizeof(float));
    hipMallocManaged(&segmentsInGPU.superbin, (maxPixelSegments )*sizeof(int));
    hipMallocManaged(&segmentsInGPU.pixelType, (maxPixelSegments )*sizeof(int));

#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&segmentsInGPU.zIns, nMemoryLocations * 7 * sizeof(float));
    hipMallocManaged(&segmentsInGPU.zLo, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.zHi, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.rtLo, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.rtHi, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.sdCut, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.dAlphaInnerMDSegmentThreshold, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.dAlphaOuterMDSegmentThreshold, nMemoryLocations * sizeof(float));
    hipMallocManaged(&segmentsInGPU.dAlphaInnerMDOuterMDThreshold, nMemoryLocations * sizeof(float));

    segmentsInGPU.zOuts = segmentsInGPU.zIns + nMemoryLocations;
    segmentsInGPU.rtIns = segmentsInGPU.zIns + nMemoryLocations * 2;
    segmentsInGPU.rtOuts = segmentsInGPU.zIns + nMemoryLocations * 3;
    segmentsInGPU.dAlphaInnerMDSegments = segmentsInGPU.zIns + nMemoryLocations * 4;
    segmentsInGPU.dAlphaOuterMDSegments = segmentsInGPU.zIns + nMemoryLocations * 5;
    segmentsInGPU.dAlphaInnerMDOuterMDs = segmentsInGPU.zIns + nMemoryLocations * 6;

#endif
#endif
    segmentsInGPU.innerLowerModuleIndices = segmentsInGPU.mdIndices + nMemoryLocations * 2;
    segmentsInGPU.outerLowerModuleIndices = segmentsInGPU.mdIndices + nMemoryLocations * 3;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 4;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 5;

    segmentsInGPU.dPhiMins = segmentsInGPU.dPhis + nMemoryLocations;
    segmentsInGPU.dPhiMaxs = segmentsInGPU.dPhis + nMemoryLocations * 2;
    segmentsInGPU.dPhiChanges = segmentsInGPU.dPhis + nMemoryLocations * 3;
    segmentsInGPU.dPhiChangeMins = segmentsInGPU.dPhis + nMemoryLocations * 4;
    segmentsInGPU.dPhiChangeMaxs = segmentsInGPU.dPhis + nMemoryLocations * 5;

    segmentsInGPU.ptIn = segmentsInGPU.dPhis + nMemoryLocations * 6;
    segmentsInGPU.ptErr = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments;
    segmentsInGPU.px = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 2;
    segmentsInGPU.py = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 3;
    segmentsInGPU.pz = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 4;
    segmentsInGPU.etaErr = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 5;
    segmentsInGPU.eta = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 6;
    segmentsInGPU.phi = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 7;
    
#pragma omp parallel for default(shared)
    for(size_t i = 0; i < nModules; i++)
    {
        segmentsInGPU.nSegments[i] = 0;
    }
}
void SDL::createSegmentsInExplicitMemory(struct segments& segmentsInGPU, unsigned int maxSegments, unsigned int nModules, unsigned int maxPixelSegments)
{
    //FIXME:Since the number of pixel segments is 10x the number of regular segments per module, we need to provide
    //extra memory to the pixel segments
    unsigned int nMemoryLocations = maxSegments * (nModules - 1) + maxPixelSegments;
#ifdef CACHE_ALLOC
    hipStream_t stream=0; 
    int dev;
    hipGetDevice(&dev);
    segmentsInGPU.mdIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations*6 *sizeof(unsigned int),stream);
    segmentsInGPU.nSegments = (unsigned int*)cms::cuda::allocate_device(dev,nModules *sizeof(unsigned int),stream);
    segmentsInGPU.dPhis = (float*)cms::cuda::allocate_device(dev,(nMemoryLocations*6 + maxPixelSegments * 8) *sizeof(float),stream);
    segmentsInGPU.superbin = (int*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int),stream);
    segmentsInGPU.pixelType = (int*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int),stream);

#else
    hipMalloc(&segmentsInGPU.mdIndices, nMemoryLocations * 6 * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.nSegments, nModules * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.dPhis, (nMemoryLocations * 6 + maxPixelSegments * 8)*sizeof(float));
    hipMalloc(&segmentsInGPU.superbin, (maxPixelSegments )*sizeof(int));
    hipMalloc(&segmentsInGPU.pixelType, (maxPixelSegments )*sizeof(int));
#endif
    hipMemset(segmentsInGPU.nSegments,0,nModules * sizeof(unsigned int));

    segmentsInGPU.innerLowerModuleIndices = segmentsInGPU.mdIndices + nMemoryLocations * 2;
    segmentsInGPU.outerLowerModuleIndices = segmentsInGPU.mdIndices + nMemoryLocations * 3;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 4;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 5;

    segmentsInGPU.dPhiMins = segmentsInGPU.dPhis + nMemoryLocations;
    segmentsInGPU.dPhiMaxs = segmentsInGPU.dPhis + nMemoryLocations * 2;
    segmentsInGPU.dPhiChanges = segmentsInGPU.dPhis + nMemoryLocations * 3;
    segmentsInGPU.dPhiChangeMins = segmentsInGPU.dPhis + nMemoryLocations * 4;
    segmentsInGPU.dPhiChangeMaxs = segmentsInGPU.dPhis + nMemoryLocations * 5;

    segmentsInGPU.ptIn = segmentsInGPU.dPhis + nMemoryLocations * 6;
    segmentsInGPU.ptErr = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments;
    segmentsInGPU.px = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 2;
    segmentsInGPU.py = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 3;
    segmentsInGPU.pz = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 4;
    segmentsInGPU.etaErr = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 5;
    segmentsInGPU.eta = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 6;
    segmentsInGPU.phi = segmentsInGPU.dPhis + nMemoryLocations * 6 + maxPixelSegments * 7;

}

SDL::segments::segments()
{
    superbin = nullptr;
    pixelType = nullptr;
    mdIndices = nullptr;
    innerLowerModuleIndices = nullptr;
    outerLowerModuleIndices = nullptr;
    innerMiniDoubletAnchorHitIndices = nullptr;
    outerMiniDoubletAnchorHitIndices = nullptr;

    nSegments = nullptr;
    dPhis = nullptr;
    dPhiMins = nullptr;
    dPhiMaxs = nullptr;
    dPhiChanges = nullptr;
    dPhiChangeMins = nullptr;
    dPhiChangeMaxs = nullptr;

#ifdef CUT_VALUE_DEBUG
    zIns = nullptr;
    zOuts = nullptr;
    rtIns = nullptr;
    rtOuts = nullptr;
    dAlphaInnerMDSegments = nullptr;
    dAlphaOuterMDSegments = nullptr;
    dAlphaInnerMDOuterMDs = nullptr;

    zLo = nullptr;
    zHi = nullptr;
    rtLo = nullptr;
    rtHi = nullptr;
    sdCut = nullptr;
    dAlphaInnerMDSegmentThreshold = nullptr;
    dAlphaOuterMDSegmentThreshold = nullptr;
    dAlphaInnerMDOuterMDThreshold = nullptr;
#endif
}

SDL::segments::~segments()
{
}

void SDL::segments::freeMemoryCache()
{
#ifdef Explicit_Seg
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,mdIndices);
    cms::cuda::free_device(dev,dPhis);
    cms::cuda::free_device(dev,nSegments);
    cms::cuda::free_device(dev,superbin);
    cms::cuda::free_device(dev,pixelType);
#else
    cms::cuda::free_managed(mdIndices);
    cms::cuda::free_managed(dPhis);
    cms::cuda::free_managed(nSegments);
    cms::cuda::free_managed(superbin);
    cms::cuda::free_managed(pixelType);
#endif
}
void SDL::segments::freeMemory()
{
    hipFree(mdIndices);
    hipFree(nSegments);
    hipFree(dPhis);
    hipFree(superbin);
    hipFree(pixelType);

#ifdef CUT_VALUE_DEBUG
    hipFree(zIns);
    hipFree(zLo);
    hipFree(zHi);
    hipFree(rtLo);
    hipFree(rtHi);
    hipFree(sdCut);
    hipFree(dAlphaInnerMDSegmentThreshold);
    hipFree(dAlphaOuterMDSegmentThreshold);
    hipFree(dAlphaInnerMDOuterMDThreshold);
#endif
}


#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addSegmentToMemory(struct segments& segmentsInGPU, unsigned int lowerMDIndex, unsigned int upperMDIndex, unsigned int innerLowerModuleIndex, unsigned int outerLowerModuleIndex, unsigned int innerMDAnchorHitIndex, unsigned int outerMDAnchorHitIndex, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, float& zIn, float& zOut, float& rtIn, float& rtOut, float& dAlphaInnerMDSegment, float& dAlphaOuterMDSegment, float&
        dAlphaInnerMDOuterMD, float& zLo, float& zHi, float& rtLo, float& rtHi, float& sdCut, float& dAlphaInnerMDSegmentThreshold, float& dAlphaOuterMDSegmentThreshold, float& dAlphaInnerMDOuterMDThreshold, unsigned int idx)
#else
__device__ void SDL::addSegmentToMemory(struct segments& segmentsInGPU, unsigned int lowerMDIndex, unsigned int upperMDIndex, unsigned int innerLowerModuleIndex, unsigned int outerLowerModuleIndex, unsigned int innerMDAnchorHitIndex, unsigned int outerMDAnchorHitIndex, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, unsigned int idx)
#endif
{
    //idx will be computed in the kernel, which is the index into which the 
    //segment will be written
    //nSegments will be incremented in the kernel

    segmentsInGPU.mdIndices[idx * 2] = lowerMDIndex;
    segmentsInGPU.mdIndices[idx * 2 + 1] = upperMDIndex;
    segmentsInGPU.innerLowerModuleIndices[idx] = innerLowerModuleIndex;
    segmentsInGPU.outerLowerModuleIndices[idx] = outerLowerModuleIndex;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices[idx] = innerMDAnchorHitIndex;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices[idx] = outerMDAnchorHitIndex;

    segmentsInGPU.dPhis[idx] = dPhi;
    segmentsInGPU.dPhiMins[idx] = dPhiMin;
    segmentsInGPU.dPhiMaxs[idx] = dPhiMax;
    segmentsInGPU.dPhiChanges[idx] = dPhiChange;
    segmentsInGPU.dPhiChangeMins[idx] = dPhiChangeMin;
    segmentsInGPU.dPhiChangeMaxs[idx] = dPhiChangeMax;

#ifdef CUT_VALUE_DEBUG
    segmentsInGPU.zIns[idx] = zIn;
    segmentsInGPU.zOuts[idx] = zOut;
    segmentsInGPU.rtIns[idx] = rtIn;
    segmentsInGPU.rtOuts[idx] = rtOut;
    segmentsInGPU.dAlphaInnerMDSegments[idx] = dAlphaInnerMDSegment;
    segmentsInGPU.dAlphaOuterMDSegments[idx] = dAlphaOuterMDSegment;
    segmentsInGPU.dAlphaInnerMDOuterMDs[idx] = dAlphaInnerMDOuterMD;

    segmentsInGPU.zLo[idx] = zLo;
    segmentsInGPU.zHi[idx] = zHi;
    segmentsInGPU.rtLo[idx] = rtLo;
    segmentsInGPU.rtHi[idx] = rtHi;
    segmentsInGPU.sdCut[idx] = sdCut;
    segmentsInGPU.dAlphaInnerMDSegmentThreshold[idx] = dAlphaInnerMDSegmentThreshold;
    segmentsInGPU.dAlphaOuterMDSegmentThreshold[idx] = dAlphaOuterMDSegmentThreshold;
    segmentsInGPU.dAlphaInnerMDOuterMDThreshold[idx] = dAlphaInnerMDOuterMDThreshold;
#endif
}

__device__ void SDL::addPixelSegmentToMemory(struct segments& segmentsInGPU, struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int innerMDIndex, unsigned int outerMDIndex, unsigned int pixelModuleIndex, unsigned int innerAnchorHitIndex, unsigned int outerAnchorHitIndex, float dPhiChange, float ptIn, float ptErr, float px, float py, float pz, float etaErr, float eta, float phi, unsigned int idx, unsigned int pixelSegmentArrayIndex, int superbin, int
        pixelType)

{
    segmentsInGPU.mdIndices[idx * 2] = innerMDIndex;
    segmentsInGPU.mdIndices[idx * 2 + 1] = outerMDIndex;
    segmentsInGPU.innerLowerModuleIndices[idx] = pixelModuleIndex;
    segmentsInGPU.outerLowerModuleIndices[idx] = pixelModuleIndex;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices[idx] = innerAnchorHitIndex;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices[idx] = outerAnchorHitIndex;
    segmentsInGPU.dPhiChanges[idx] = dPhiChange;
    segmentsInGPU.ptIn[pixelSegmentArrayIndex] = ptIn;
    segmentsInGPU.ptErr[pixelSegmentArrayIndex] = ptErr;
    segmentsInGPU.px[pixelSegmentArrayIndex] = px;
    segmentsInGPU.py[pixelSegmentArrayIndex] = py;
    segmentsInGPU.pz[pixelSegmentArrayIndex] = pz;
    segmentsInGPU.etaErr[pixelSegmentArrayIndex] = etaErr;
    segmentsInGPU.eta[pixelSegmentArrayIndex] = eta;
    segmentsInGPU.phi[pixelSegmentArrayIndex] = phi;

    segmentsInGPU.superbin[pixelSegmentArrayIndex] = superbin;
    segmentsInGPU.pixelType[pixelSegmentArrayIndex] = pixelType;
}

__device__ void SDL::dAlphaThreshold(float* dAlphaThresholdValues, struct hits& hitsInGPU, struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, unsigned int& innerMiniDoubletAnchorHitIndex, unsigned int& outerMiniDoubletAnchorHitIndex, unsigned int& innerLowerModuleIndex, unsigned int& outerLowerModuleIndex, unsigned int& innerMDIndex, unsigned int& outerMDIndex)
{
    float sdMuls = (modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel) ? miniMulsPtScaleBarrel[modulesInGPU.layers[innerLowerModuleIndex]-1] * 3.f/ptCut : miniMulsPtScaleEndcap[modulesInGPU.layers[innerLowerModuleIndex]-1] * 3.f/ptCut;

    // BField dAlpha

    float innerMiniDoubletAnchorHitRt = hitsInGPU.rts[innerMiniDoubletAnchorHitIndex];
    float outerMiniDoubletAnchorHitRt = hitsInGPU.rts[outerMiniDoubletAnchorHitIndex];
    float innerMiniDoubletAnchorHitZ = hitsInGPU.zs[innerMiniDoubletAnchorHitIndex];
    float outerMiniDoubletAnchorHitZ = hitsInGPU.zs[outerMiniDoubletAnchorHitIndex];

    //more accurate then outer rt - inner rt
    float segmentY = hitsInGPU.ys[outerMiniDoubletAnchorHitIndex] - hitsInGPU.ys[innerMiniDoubletAnchorHitIndex];
    float segmentX = hitsInGPU.xs[outerMiniDoubletAnchorHitIndex]- hitsInGPU.xs[innerMiniDoubletAnchorHitIndex]; 
    float segmentDr = sqrtf((segmentY * segmentY) + (segmentX * segmentX));
    

    const float dAlpha_Bfield = asinf(fminf(segmentDr * k2Rinv1GeVf/ptCut, sinAlphaMax));

    bool isInnerTilted = modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel and modulesInGPU.sides[innerLowerModuleIndex] != SDL::Center;
    bool isOuterTilted = modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Barrel and modulesInGPU.sides[outerLowerModuleIndex] != SDL::Center;
    float drdzInner = -1.f;
    float drdzOuter = -1.f;
    if(isInnerTilted)
    {
        if(modulesInGPU.moduleLayerType[innerLowerModuleIndex] == Strip)
        {
            drdzInner = modulesInGPU.drdzs[innerLowerModuleIndex];
        }
        else
        {
            drdzInner = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(innerLowerModuleIndex)];
        }
    }
    if(isOuterTilted)
    {
        if(modulesInGPU.moduleLayerType[outerLowerModuleIndex] == Strip)
        {
            drdzOuter = modulesInGPU.drdzs[outerLowerModuleIndex];
        }
        else
        {
            drdzOuter = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(outerLowerModuleIndex)];
        }

    }

    float innerModuleGapSize = SDL::moduleGapSize(modulesInGPU, innerLowerModuleIndex);
    float outerModuleGapSize = SDL::moduleGapSize(modulesInGPU, outerLowerModuleIndex);
    const float innerminiTilt = isInnerTilted ? (0.5f * pixelPSZpitch * drdzInner / sqrtf(1.f + drdzInner * drdzInner) / innerModuleGapSize) : 0;

    const float outerminiTilt = isOuterTilted ? (0.5f * pixelPSZpitch * drdzOuter / sqrtf(1.f + drdzOuter * drdzOuter) / outerModuleGapSize) : 0;

    float miniDelta = innerModuleGapSize; 
 

    float sdLumForInnerMini;    
    float sdLumForOuterMini;

    if (modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel)
    {
        sdLumForInnerMini = innerminiTilt * dAlpha_Bfield;
    }
    else
    {
        sdLumForInnerMini = mdsInGPU.dphis[innerMDIndex] * 15.0f / mdsInGPU.dzs[innerMDIndex];
    }

    if (modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Barrel)
    {
        sdLumForOuterMini = outerminiTilt * dAlpha_Bfield;
    }
    else
    {
        sdLumForOuterMini = mdsInGPU.dphis[outerMDIndex] * 15.0f / mdsInGPU.dzs[outerMDIndex];
    }


    //Unique stuff for the segment dudes alone

    float dAlpha_res_inner = 0.02f/miniDelta * (modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel ? 1.0f : fabsf(innerMiniDoubletAnchorHitZ/innerMiniDoubletAnchorHitRt));
    float dAlpha_res_outer = 0.02f/miniDelta * (modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Barrel ? 1.0f : fabsf(outerMiniDoubletAnchorHitZ/outerMiniDoubletAnchorHitRt));

 
    float dAlpha_res = dAlpha_res_inner + dAlpha_res_outer;

    if (modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel and modulesInGPU.sides[innerLowerModuleIndex] == SDL::Center)
    {
        dAlphaThresholdValues[0] = dAlpha_Bfield + sqrt(dAlpha_res * dAlpha_res + sdMuls * sdMuls);       
    }
    else
    {
        dAlphaThresholdValues[0] = dAlpha_Bfield + sqrt(dAlpha_res * dAlpha_res + sdMuls * sdMuls + sdLumForInnerMini * sdLumForInnerMini);    
    }

    if(modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Barrel and modulesInGPU.sides[outerLowerModuleIndex] == SDL::Center)
    {
        dAlphaThresholdValues[1] = dAlpha_Bfield + sqrt(dAlpha_res * dAlpha_res + sdMuls * sdMuls);    
    }
    else
    {
        dAlphaThresholdValues[1] = dAlpha_Bfield + sqrt(dAlpha_res * dAlpha_res + sdMuls * sdMuls + sdLumForOuterMini * sdLumForOuterMini);
    }

    //Inner to outer 
    dAlphaThresholdValues[2] = dAlpha_Bfield + sqrt(dAlpha_res * dAlpha_res + sdMuls * sdMuls);

}


__device__ bool SDL::runSegmentDefaultAlgoEndcap(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, unsigned int& innerLowerModuleIndex, unsigned int& outerLowerModuleIndex, unsigned int& innerMDIndex, unsigned int& outerMDIndex, float& zIn, float& zOut, float& rtIn, float& rtOut, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, float& dAlphaInnerMDSegment, float& dAlphaOuterMDSegment,
        float& rtLo, float& rtHi, float& sdCut, float& dAlphaInnerMDSegmentThreshold, float& dAlphaOuterMDSegmentThreshold, float& dAlphaInnerMDOuterMDThreshold, float&
        dAlphaInnerMDOuterMD, unsigned int& innerMiniDoubletAnchorHitIndex, unsigned int& outerMiniDoubletAnchorHitIndex)
{
    bool pass = true;
    
    if(mdsInGPU.pixelModuleFlag[innerMDIndex] >= 0)
    {
        if(mdsInGPU.pixelModuleFlag[innerMDIndex] == 0)
        {    
            innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2]; 
        }
        else
        {
            innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2 + 1];
        }
    }
    else
    {
        innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2];
    }

    if(mdsInGPU.pixelModuleFlag[outerMDIndex] >= 0)
    {
        if(mdsInGPU.pixelModuleFlag[outerMDIndex] == 0)
        {    
            outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2]; 
        }
        else
        {
            outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2 + 1];
        }
    }
    else
    {
        outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2];
    }

    rtIn = hitsInGPU.rts[innerMiniDoubletAnchorHitIndex];
    rtOut = hitsInGPU.rts[outerMiniDoubletAnchorHitIndex];
    zIn = hitsInGPU.zs[innerMiniDoubletAnchorHitIndex];
    zOut = hitsInGPU.zs[outerMiniDoubletAnchorHitIndex];

    bool outerLayerEndcapTwoS = (modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Endcap) and (modulesInGPU.moduleType[outerLowerModuleIndex] == SDL::TwoS);

    
    float sdSlope = asinf(fminf(rtOut * k2Rinv1GeVf / ptCut, sinAlphaMax));
    float sdPVoff = 0.1/rtOut;
    float disks2SMinRadius = 60.f;

    float rtGeom =  ((rtIn < disks2SMinRadius && rtOut < disks2SMinRadius) ? (2.f * pixelPSZpitch)
            : ((rtIn < disks2SMinRadius || rtOut < disks2SMinRadius) ? (pixelPSZpitch + strip2SZpitch)
            : (2.f * strip2SZpitch)));


    //cut 0 - z compatibility
    if(zIn * zOut < 0)
    {
        pass = false;
    }

    float dz = zOut - zIn;
    float dLum = copysignf(deltaZLum, zIn);
    float drtDzScale = sdSlope/tanf(sdSlope);

    rtLo = fmaxf(rtIn * (1.f + dz / (zIn + dLum) * drtDzScale) - rtGeom,  rtIn - 0.5f * rtGeom); //rt should increase
    rtHi = rtIn * (zOut - dLum) / (zIn - dLum) + rtGeom; //dLum for luminous; rGeom for measurement size; no tanTheta_loc(pt) correction

    //completeness

    if(not(rtOut >= rtLo and rtOut <= rtHi))
    {
        pass = false;
    }

    dPhi = deltaPhi(hitsInGPU.xs[innerMiniDoubletAnchorHitIndex], hitsInGPU.ys[innerMiniDoubletAnchorHitIndex], hitsInGPU.zs[innerMiniDoubletAnchorHitIndex], hitsInGPU.xs[outerMiniDoubletAnchorHitIndex], hitsInGPU.ys[outerMiniDoubletAnchorHitIndex], hitsInGPU.zs[outerMiniDoubletAnchorHitIndex]);

    sdCut = sdSlope;
    unsigned int outerEdgeIndex;
    if(outerLayerEndcapTwoS)
    {
        outerEdgeIndex = outerMiniDoubletAnchorHitIndex;

        float dPhiPos_high = deltaPhi(hitsInGPU.xs[innerMiniDoubletAnchorHitIndex], hitsInGPU.ys[innerMiniDoubletAnchorHitIndex], hitsInGPU.zs[innerMiniDoubletAnchorHitIndex], hitsInGPU.highEdgeXs[outerEdgeIndex], hitsInGPU.highEdgeYs[outerEdgeIndex], hitsInGPU.zs[outerMiniDoubletAnchorHitIndex]);

        float dPhiPos_low = deltaPhi(hitsInGPU.xs[innerMiniDoubletAnchorHitIndex], hitsInGPU.ys[innerMiniDoubletAnchorHitIndex], hitsInGPU.zs[innerMiniDoubletAnchorHitIndex], hitsInGPU.lowEdgeXs[outerEdgeIndex], hitsInGPU.lowEdgeYs[outerEdgeIndex], hitsInGPU.zs[outerMiniDoubletAnchorHitIndex]);

        dPhiMax = fabsf(dPhiPos_high) > fabsf(dPhiPos_low) ? dPhiPos_high : dPhiPos_low;
        dPhiMin = fabsf(dPhiPos_high) > fabsf(dPhiPos_low) ? dPhiPos_low : dPhiPos_high;
    }
    else
    {
        dPhiMax = dPhi;
        dPhiMin = dPhi;
    }

    if(fabsf(dPhi) > sdCut)
    {
        pass = false;
    }

    float dzFrac = dz/zIn;
    dPhiChange = dPhi/dzFrac * (1.f + dzFrac);
    dPhiChangeMin = dPhiMin/dzFrac * (1.f + dzFrac);
    dPhiChangeMax = dPhiMax/dzFrac * (1.f + dzFrac);

    if(fabsf(dPhiChange) > sdCut)
    {
        pass = false;
    }



    float dAlphaThresholdValues[3];
    dAlphaThreshold(dAlphaThresholdValues, hitsInGPU, modulesInGPU, mdsInGPU, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex);

    dAlphaInnerMDSegmentThreshold = dAlphaThresholdValues[0];
    dAlphaOuterMDSegmentThreshold = dAlphaThresholdValues[1];
    dAlphaInnerMDOuterMDThreshold = dAlphaThresholdValues[2];

    float innerMDAlpha = mdsInGPU.dphichanges[innerMDIndex];
    float outerMDAlpha = mdsInGPU.dphichanges[outerMDIndex];
    dAlphaInnerMDSegment = innerMDAlpha - dPhiChange;
    dAlphaOuterMDSegment = outerMDAlpha - dPhiChange;
    dAlphaInnerMDOuterMD = innerMDAlpha - outerMDAlpha;
   
 
    if(fabsf(dAlphaInnerMDSegment) >= dAlphaThresholdValues[0])
    {
        pass = false;
    }

    if(fabsf(dAlphaOuterMDSegment) >= dAlphaThresholdValues[1])
    {
        pass = false;
    }

    if(fabsf(dAlphaInnerMDOuterMD) >= dAlphaThresholdValues[2])
    {
        pass = false;
    }


    return pass;
}

__device__ bool SDL::runSegmentDefaultAlgoBarrel(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, unsigned int& innerLowerModuleIndex, unsigned int& outerLowerModuleIndex, unsigned int& innerMDIndex, unsigned int& outerMDIndex, float& zIn, float& zOut, float& rtIn, float& rtOut, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, float& dAlphaInnerMDSegment, float& dAlphaOuterMDSegment, float&
        dAlphaInnerMDOuterMD, float& zLo, float& zHi, float& sdCut, float& dAlphaInnerMDSegmentThreshold, float& dAlphaOuterMDSegmentThreshold, float& dAlphaInnerMDOuterMDThreshold, unsigned int& innerMiniDoubletAnchorHitIndex, unsigned int& outerMiniDoubletAnchorHitIndex)
{
    bool pass = true;
   
    float sdMuls = (modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel) ? miniMulsPtScaleBarrel[modulesInGPU.layers[innerLowerModuleIndex]-1] * 3.f/ptCut : miniMulsPtScaleEndcap[modulesInGPU.layers[innerLowerModuleIndex]-1] * 3.f/ptCut;


    if(mdsInGPU.pixelModuleFlag[innerMDIndex] >= 0)
    {
        if(mdsInGPU.pixelModuleFlag[innerMDIndex] == 0)
        {    
            innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2];
        }
        else
        {
            innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2 + 1]; 
        }
    }
    else
    {
        innerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[innerMDIndex * 2];
    }

    if(mdsInGPU.pixelModuleFlag[outerMDIndex] >= 0)
    {
        if(mdsInGPU.pixelModuleFlag[outerMDIndex] == 0)
        {    
            outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2];
 		
        }
        else
        {
            outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2 + 1];
	    
        }
    }
    else
    {
        outerMiniDoubletAnchorHitIndex = mdsInGPU.hitIndices[outerMDIndex * 2];
    }


    rtIn = hitsInGPU.rts[innerMiniDoubletAnchorHitIndex];
    rtOut = hitsInGPU.rts[outerMiniDoubletAnchorHitIndex];
    zIn = hitsInGPU.zs[innerMiniDoubletAnchorHitIndex];
    zOut = hitsInGPU.zs[outerMiniDoubletAnchorHitIndex];

    float sdSlope = asinf(fminf(rtOut * k2Rinv1GeVf / ptCut, sinAlphaMax));
    float sdPVoff = 0.1f/rtOut;
    float dzDrtScale = tanf(sdSlope)/sdSlope; //FIXME: need appropriate value

    const float zGeom = modulesInGPU.layers[innerLowerModuleIndex] <= 2 ? 2.f * pixelPSZpitch : 2.f * strip2SZpitch;

    zLo = zIn + (zIn - deltaZLum) * (rtOut / rtIn - 1.f) * (zIn > 0.f ? 1.f : dzDrtScale) - zGeom; //slope-correction only on outer end
    zHi = zIn + (zIn + deltaZLum) * (rtOut / rtIn - 1.f) * (zIn < 0.f ? 1.f : dzDrtScale) + zGeom;

    //cut 1 - z compatibility
    if(not(zOut >= zLo and zOut <= zHi))
    {
        pass = false;
    }


    dPhi = deltaPhi(hitsInGPU.xs[innerMiniDoubletAnchorHitIndex], hitsInGPU.ys[innerMiniDoubletAnchorHitIndex], hitsInGPU.zs[innerMiniDoubletAnchorHitIndex], hitsInGPU.xs[outerMiniDoubletAnchorHitIndex], hitsInGPU.ys[outerMiniDoubletAnchorHitIndex], hitsInGPU.zs[outerMiniDoubletAnchorHitIndex]);
    sdCut = sdSlope + sqrtf(sdMuls * sdMuls + sdPVoff * sdPVoff);

    if(not( fabsf(dPhi) <= sdCut ))
    {
        pass = false;
    }

    dPhiChange = deltaPhiChange(hitsInGPU.xs[innerMiniDoubletAnchorHitIndex], hitsInGPU.ys[innerMiniDoubletAnchorHitIndex], hitsInGPU.zs[innerMiniDoubletAnchorHitIndex], hitsInGPU.xs[outerMiniDoubletAnchorHitIndex], hitsInGPU.ys[outerMiniDoubletAnchorHitIndex], hitsInGPU.zs[outerMiniDoubletAnchorHitIndex]);

    if(not( fabsf(dPhiChange) <= sdCut ))
    {
        pass = false;
    }
    
    float dAlphaThresholdValues[3];
    dAlphaThreshold(dAlphaThresholdValues, hitsInGPU, modulesInGPU, mdsInGPU, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex);

    float innerMDAlpha = mdsInGPU.dphichanges[innerMDIndex];
    float outerMDAlpha = mdsInGPU.dphichanges[outerMDIndex];
    dAlphaInnerMDSegment = innerMDAlpha - dPhiChange;
    dAlphaOuterMDSegment = outerMDAlpha - dPhiChange;
    dAlphaInnerMDOuterMD = innerMDAlpha - outerMDAlpha;

    dAlphaInnerMDSegmentThreshold = dAlphaThresholdValues[0];
    dAlphaOuterMDSegmentThreshold = dAlphaThresholdValues[1];
    dAlphaInnerMDOuterMDThreshold = dAlphaThresholdValues[2];
    
    if(fabsf(dAlphaInnerMDSegment) >= dAlphaThresholdValues[0])
    {
        pass = false;
    }

    if(fabsf(dAlphaOuterMDSegment) >= dAlphaThresholdValues[1])
    {
        pass = false;
    }

    if(fabsf(dAlphaInnerMDOuterMD) >= dAlphaThresholdValues[2])
    {
        pass = false;
    }

    return pass;
}

__device__ bool SDL::runSegmentDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, unsigned int& innerLowerModuleIndex, unsigned int& outerLowerModuleIndex, unsigned int& innerMDIndex, unsigned int& outerMDIndex, float& zIn, float& zOut, float& rtIn, float& rtOut, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, float& dAlphaInnerMDSegment, float& dAlphaOuterMDSegment, float&
        dAlphaInnerMDOuterMD, float& zLo, float& zHi, float& rtLo, float& rtHi, float& sdCut, float& dAlphaInnerMDSegmentThreshold, float& dAlphaOuterMDSegmentThreshold, float& dAlphaInnerMDOuterMDThreshold, unsigned int& innerMiniDoubletAnchorHitIndex, unsigned int& outerMiniDoubletAnchorHitIndex)
{
    zLo = -999;
    zHi = -999;
    rtLo = -999;
    rtHi = -999;

    bool pass = true;

    if(modulesInGPU.subdets[innerLowerModuleIndex] == SDL::Barrel)
    {
        if(modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Barrel)
        {
            pass = runSegmentDefaultAlgoBarrel(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);
        }
        else
        {
            pass = runSegmentDefaultAlgoEndcap(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

        }
    }  

    else
    {
        if(modulesInGPU.subdets[outerLowerModuleIndex] == SDL::Endcap)
            pass = runSegmentDefaultAlgoEndcap(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);
        else
            pass = runSegmentDefaultAlgoBarrel(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    }

    return pass;
}
void SDL::printSegment(struct SDL::segments& segmentsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::hits& hitsInGPU, struct SDL::modules& modulesInGPU, unsigned int segmentIndex)
{
    unsigned int innerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2];
    unsigned int outerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2 + 1];
    std::cout<<std::endl;
    std::cout<<"sg_dPhiChange : "<<segmentsInGPU.dPhiChanges[segmentIndex] << std::endl<<std::endl;

    std::cout << "Inner Mini-Doublet" << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex);
    }
    std::cout<<std::endl<<" Outer Mini-Doublet" <<std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, outerMDIndex);
    }
}
