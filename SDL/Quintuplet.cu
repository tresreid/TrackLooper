#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
#include "allocate.h"

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    innerRadius = nullptr;
    outerRadius = nullptr;
    isDup = nullptr;
    pt = nullptr;
    layer = nullptr;
//    eta = nullptr;
//    phi = nullptr;

#ifdef CUT_VALUE_DEBUG
    innerRadiusMin = nullptr;
    innerRadiusMin2S = nullptr;
    innerRadiusMax = nullptr;
    innerRadiusMax2S = nullptr;
    bridgeRadius = nullptr;
    bridgeRadiusMin = nullptr;
    bridgeRadiusMin2S = nullptr;
    bridgeRadiusMax = nullptr;
    bridgeRadiusMax2S = nullptr;
    outerRadiusMin = nullptr;
    outerRadiusMin2S = nullptr;
    outerRadiusMax = nullptr;
    outerRadiusMax2S = nullptr;
#endif
}

SDL::quintuplets::~quintuplets()
{
}

void SDL::quintuplets::freeMemoryCache()
{
#ifdef Explicit_T5
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, nQuintuplets);
    cms::cuda::free_device(dev, innerRadius);
    cms::cuda::free_device(dev, outerRadius);
    cms::cuda::free_device(dev, isDup);
    cms::cuda::free_device(dev, pt);
#else
    cms::cuda::free_managed(tripletIndices);
    cms::cuda::free_managed(lowerModuleIndices);
    cms::cuda::free_managed(nQuintuplets);
    cms::cuda::free_managed(innerRadius);
    cms::cuda::free_managed(outerRadius);
    cms::cuda::free_managed(isDup);
    cms::cuda::free_managed(pt);
#endif
}

void SDL::quintuplets::freeMemory()
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(innerRadius);
    hipFree(outerRadius);
    hipFree(isDup);
    hipFree(pt);
    hipFree(layer);

#ifdef CUT_VALUE_DEBUG
    hipFree(innerRadiusMin);
    hipFree(innerRadiusMin2S);
    hipFree(innerRadiusMax);
    hipFree(innerRadiusMax2S);
    hipFree(bridgeRadius);
    hipFree(bridgeRadiusMin);
    hipFree(bridgeRadiusMin2S);
    hipFree(bridgeRadiusMax);
    hipFree(bridgeRadiusMax2S);
    hipFree(outerRadiusMin);
    hipFree(outerRadiusMin2S);
    hipFree(outerRadiusMax);
    hipFree(outerRadiusMax2S);
#endif
}

//TODO:Reuse the track candidate one instead of this!
void SDL::createEligibleModulesListForQuintuplets(struct modules& modulesInGPU,struct triplets& tripletsInGPU, unsigned int& nEligibleModules, unsigned int* indicesOfEligibleModules, unsigned int maxQuintuplets, unsigned int& maxTriplets)
{
    unsigned int nLowerModules;
    maxTriplets = 0;
    hipMemcpy(&nLowerModules,modulesInGPU.nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU.nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemset(modulesInGPU.quintupletModuleIndices, -1, sizeof(int) * (nLowerModules));

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU.subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, nLowerModules * sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU.lowerModuleIndices, nLowerModules * sizeof(unsigned int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU.layers,nModules * sizeof(short),hipMemcpyDeviceToHost);
    int* module_quintupletModuleIndices;
    hipHostMalloc(&module_quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpy(module_quintupletModuleIndices,modulesInGPU.quintupletModuleIndices,nLowerModules *sizeof(int),hipMemcpyDeviceToHost);

    unsigned int* nTriplets;
    hipHostMalloc(&nTriplets, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU.nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);

    //start filling
    for(unsigned int i = 0; i < nLowerModules; i++)
    {
        //condition for a quintuple to exist for a module
        //TCs don't exist for layers 5 and 6 barrel, and layers 2,3,4,5 endcap
        unsigned int idx = module_lowerModuleIndices[i];
        if(((module_subdets[idx] == SDL::Barrel and module_layers[idx] < 5) or (module_subdets[idx] == SDL::Endcap and module_layers[idx] == 1)) and nTriplets[i] != 0)
        {
            module_quintupletModuleIndices[i] = nEligibleModules * maxQuintuplets;
            indicesOfEligibleModules[nEligibleModules] = i;
            nEligibleModules++;
            maxTriplets = max(nTriplets[i], maxTriplets);

        }
    }
    hipMemcpy(modulesInGPU.quintupletModuleIndices,module_quintupletModuleIndices,nLowerModules*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.nEligibleT5Modules,&nEligibleModules,sizeof(unsigned int),hipMemcpyHostToDevice);
    hipHostFree(module_subdets);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_layers);
    hipHostFree(module_quintupletModuleIndices);
    hipHostFree(nTriplets);
}


void SDL::createQuintupletsInUnifiedMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& maxQuintuplets, const unsigned int& nLowerModules, const unsigned int& nEligibleModules)
{
    unsigned int nMemoryLocations = maxQuintuplets * nEligibleModules;
    std::cout<<"Number of eligible T5 modules = "<<nEligibleModules<<std::endl;

#ifdef CACHE_ALLOC
    hipStream_t stream = 0;
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_managed(nMemoryLocations * 2 * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_managed(nMemoryLocations * 5 * sizeof(unsigned int), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_managed(nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (float*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.outerRadius = (float*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.pt = (float*)cms::cuda::allocate_managed(nMemoryLocations *5* sizeof(float), stream);
#else
    hipMallocManaged(&quintupletsInGPU.tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.lowerModuleIndices, 5 * nMemoryLocations * sizeof(unsigned int));

    hipMallocManaged(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.innerRadius, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadius, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.isDup, nMemoryLocations * sizeof(bool));
    hipMallocManaged(&quintupletsInGPU.pt, nMemoryLocations *16* sizeof(float));
    hipMallocManaged(&quintupletsInGPU.layer, nMemoryLocations *1* sizeof(int));

#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadius, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax2S, nMemoryLocations * sizeof(float));
#endif
#endif
    quintupletsInGPU.eta = quintupletsInGPU.pt + nMemoryLocations;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nMemoryLocations;
    quintupletsInGPU.distance = quintupletsInGPU.pt + 3*nMemoryLocations;
    quintupletsInGPU.slope = quintupletsInGPU.pt + 4*nMemoryLocations;
    quintupletsInGPU.score = quintupletsInGPU.pt + 5*nMemoryLocations;
    quintupletsInGPU.score2 = quintupletsInGPU.pt + 6*nMemoryLocations;
    quintupletsInGPU.score3 = quintupletsInGPU.pt + 7*nMemoryLocations;
    quintupletsInGPU.score4 = quintupletsInGPU.pt + 8*nMemoryLocations;
    quintupletsInGPU.score5 = quintupletsInGPU.pt + 9*nMemoryLocations;
    quintupletsInGPU.score6 = quintupletsInGPU.pt + 10*nMemoryLocations;
    quintupletsInGPU.score7 = quintupletsInGPU.pt + 11*nMemoryLocations;
    quintupletsInGPU.score8 = quintupletsInGPU.pt + 12*nMemoryLocations;
    quintupletsInGPU.score9 = quintupletsInGPU.pt + 13*nMemoryLocations;
    quintupletsInGPU.p1 = quintupletsInGPU.pt + 14*nMemoryLocations;
    quintupletsInGPU.p2 = quintupletsInGPU.pt + 15*nMemoryLocations;
#pragma omp parallel for
    for(size_t i = 0; i<nLowerModules;i++)
    {
        quintupletsInGPU.nQuintuplets[i] = 0;
    }

}

void SDL::createQuintupletsInExplicitMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& maxQuintuplets, const unsigned int& nLowerModules, const unsigned int& nEligibleModules)
{
    unsigned int nMemoryLocations = nEligibleModules * maxQuintuplets;
#ifdef CACHE_ALLOC
    hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_device(dev, 2 * nMemoryLocations * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev, 5 * nMemoryLocations * sizeof(unsigned int), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.outerRadius = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.pt = (float*)cms::cuda::allocate_device(dev, nMemoryLocations *5* sizeof(float), stream);
#else
    hipMalloc(&quintupletsInGPU.tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.lowerModuleIndices, 5 * nMemoryLocations * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.innerRadius, nMemoryLocations * sizeof(float));
    hipMalloc(&quintupletsInGPU.outerRadius, nMemoryLocations * sizeof(float));
    hipMalloc(&quintupletsInGPU.isDup, nMemoryLocations * sizeof(bool));
    hipMalloc(&quintupletsInGPU.pt, nMemoryLocations *16* sizeof(float));
    hipMalloc(&quintupletsInGPU.layer, nMemoryLocations *1* sizeof(int));
#endif
    quintupletsInGPU.eta = quintupletsInGPU.pt + nMemoryLocations;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nMemoryLocations;
    quintupletsInGPU.distance = quintupletsInGPU.pt + 3*nMemoryLocations;
    quintupletsInGPU.slope = quintupletsInGPU.pt + 4*nMemoryLocations;
    quintupletsInGPU.score = quintupletsInGPU.pt + 5*nMemoryLocations;
    quintupletsInGPU.score2 = quintupletsInGPU.pt + 6*nMemoryLocations;
    quintupletsInGPU.score3 = quintupletsInGPU.pt + 7*nMemoryLocations;
    quintupletsInGPU.score4 = quintupletsInGPU.pt + 8*nMemoryLocations;
    quintupletsInGPU.score5 = quintupletsInGPU.pt + 9*nMemoryLocations;
    quintupletsInGPU.score6 = quintupletsInGPU.pt + 10*nMemoryLocations;
    quintupletsInGPU.score7 = quintupletsInGPU.pt + 11*nMemoryLocations;
    quintupletsInGPU.score8 = quintupletsInGPU.pt + 12*nMemoryLocations;
    quintupletsInGPU.score9 = quintupletsInGPU.pt + 13*nMemoryLocations;
    quintupletsInGPU.p1 = quintupletsInGPU.pt + 14*nMemoryLocations;
    quintupletsInGPU.p2 = quintupletsInGPU.pt + 15*nMemoryLocations;
    hipMemset(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(unsigned int));
}


#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, unsigned int lowerModule1, unsigned int lowerModule2, unsigned int lowerModule3, unsigned int lowerModule4, unsigned int lowerModule5, float innerRadius, float innerRadiusMin, float innerRadiusMax, float outerRadius, float outerRadiusMin, float outerRadiusMax, float bridgeRadius, float bridgeRadiusMin, float bridgeRadiusMax,
        float innerRadiusMin2S, float innerRadiusMax2S, float bridgeRadiusMin2S, float bridgeRadiusMax2S, float outerRadiusMin2S, float outerRadiusMax2S,unsigned int quintupletIndex)

#else
__device__ void SDL::addQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, unsigned int lowerModule1, unsigned int lowerModule2, unsigned int lowerModule3, unsigned int lowerModule4, unsigned int lowerModule5, float innerRadius, float outerRadius, unsigned int quintupletIndex,bool isDup, float pt, float eta, float phi,float distance,float* scores,int layer)
#endif

{
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = innerTripletIndex;
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;
    quintupletsInGPU.innerRadius[quintupletIndex] = innerRadius;
    quintupletsInGPU.outerRadius[quintupletIndex] = outerRadius;
    quintupletsInGPU.isDup[quintupletIndex] = isDup;
    quintupletsInGPU.pt[quintupletIndex] = pt;
    quintupletsInGPU.eta[quintupletIndex] = eta;
    quintupletsInGPU.phi[quintupletIndex] = phi;
    quintupletsInGPU.distance[quintupletIndex] = distance;
    quintupletsInGPU.slope[quintupletIndex] = scores[0];
    quintupletsInGPU.score[quintupletIndex] = scores[1];
    quintupletsInGPU.score2[quintupletIndex] = scores[2];
    quintupletsInGPU.score3[quintupletIndex] = scores[3];
    quintupletsInGPU.score4[quintupletIndex] = scores[4];
    quintupletsInGPU.score5[quintupletIndex] = scores[5];
    quintupletsInGPU.score6[quintupletIndex] = scores[6];
    quintupletsInGPU.score7[quintupletIndex] = scores[7];
    quintupletsInGPU.score8[quintupletIndex] = scores[8];
    quintupletsInGPU.score9[quintupletIndex] = scores[9];
    quintupletsInGPU.p1[quintupletIndex] = scores[10];
    quintupletsInGPU.p2[quintupletIndex] = scores[11];
    quintupletsInGPU.layer[quintupletIndex] = layer;

#ifdef CUT_VALUE_DEBUG
    quintupletsInGPU.innerRadiusMin[quintupletIndex] = innerRadiusMin;
    quintupletsInGPU.innerRadiusMax[quintupletIndex] = innerRadiusMax;
    quintupletsInGPU.outerRadiusMin[quintupletIndex] = outerRadiusMin;
    quintupletsInGPU.outerRadiusMax[quintupletIndex] = outerRadiusMax;
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.bridgeRadiusMin[quintupletIndex] = bridgeRadiusMin;
    quintupletsInGPU.bridgeRadiusMax[quintupletIndex] = bridgeRadiusMax;
    quintupletsInGPU.innerRadiusMin2S[quintupletIndex] = innerRadiusMin2S;
    quintupletsInGPU.innerRadiusMax2S[quintupletIndex] = innerRadiusMax2S;
    quintupletsInGPU.bridgeRadiusMin2S[quintupletIndex] = bridgeRadiusMin2S;
    quintupletsInGPU.bridgeRadiusMax2S[quintupletIndex] = bridgeRadiusMax2S;
    quintupletsInGPU.outerRadiusMin2S[quintupletIndex] = outerRadiusMin2S;
    quintupletsInGPU.outerRadiusMax2S[quintupletIndex] = outerRadiusMax2S;
#endif

}
__device__ void SDL::rmQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU,unsigned int quintupletIndex)
{
    quintupletsInGPU.isDup[quintupletIndex] = 1;

}
__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, unsigned int innerTripletIndex, unsigned int outerTripletIndex, float& innerRadius, float& innerRadiusMin, float&
    innerRadiusMax, float& outerRadius, float& outerRadiusMin, float& outerRadiusMax, float& bridgeRadius, float& bridgeRadiusMin, float& bridgeRadiusMax, float& innerRadiusMin2S, float& innerRadiusMax2S, float& bridgeRadiusMin2S, float& bridgeRadiusMax2S, float& outerRadiusMin2S, float& outerRadiusMax2S)
{
    bool pass = true;

    //if(not T5HasCommonMiniDoublet(tripletsInGPU, segmentsInGPU, innerTripletIndex, outerTripletIndex))
    //{
    //    pass = false;
    //}

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];

    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex]; //outer triplet inner segmnet inner MD index

    if (innerOuterOuterMiniDoubletIndex != outerInnerInnerMiniDoubletIndex) pass = false;

    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[thirdSegmentIndex], segmentsInGPU.outerLowerModuleIndices[thirdSegmentIndex], firstSegmentIndex, thirdSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ))
    {
        pass = false;
    }
    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[fourthSegmentIndex], segmentsInGPU.outerLowerModuleIndices[fourthSegmentIndex], firstSegmentIndex, fourthSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ))
    {
        pass = false;
    }

    //radius computation from the three triplet MD anchor hits
    unsigned int innerTripletFirstSegmentAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int innerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int innerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int outerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int outerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];

    float x1 = hitsInGPU.xs[innerTripletFirstSegmentAnchorHitIndex];
    float x2 = hitsInGPU.xs[innerTripletSecondSegmentAnchorHitIndex];
    float x3 = hitsInGPU.xs[innerTripletThirdSegmentAnchorHitIndex];
    float x4 = hitsInGPU.xs[outerTripletSecondSegmentAnchorHitIndex];
    float x5 = hitsInGPU.xs[outerTripletThirdSegmentAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerTripletFirstSegmentAnchorHitIndex];
    float y2 = hitsInGPU.ys[innerTripletSecondSegmentAnchorHitIndex];
    float y3 = hitsInGPU.ys[innerTripletThirdSegmentAnchorHitIndex];
    float y4 = hitsInGPU.ys[outerTripletSecondSegmentAnchorHitIndex];
    float y5 = hitsInGPU.ys[outerTripletThirdSegmentAnchorHitIndex];


    //construct the arrays
    float x1Vec[] = {x1, x1, x1};
    float y1Vec[] = {y1, y1, y1};
    float x2Vec[] = {x2, x2, x2};
    float y2Vec[] = {y2, y2, y2};
    float x3Vec[] = {x3, x3, x3};
    float y3Vec[] = {y3, y3, y3};
    //float x4Vec[] = {x4, x4, x4};
    //float y4Vec[] = {y4, y4, y4};
    //float x5Vec[] = {x5, x5, x5};
    //float y5Vec[] = {y5, y5, y5};

    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS)
    {
        x1Vec[1] = hitsInGPU.lowEdgeXs[innerTripletFirstSegmentAnchorHitIndex];
        x1Vec[2] = hitsInGPU.highEdgeXs[innerTripletFirstSegmentAnchorHitIndex];

        y1Vec[1] = hitsInGPU.lowEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
        y1Vec[2] = hitsInGPU.highEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
    }
    if(modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS)
    {
        x2Vec[1] = hitsInGPU.lowEdgeXs[innerTripletSecondSegmentAnchorHitIndex];
        x2Vec[2] = hitsInGPU.highEdgeXs[innerTripletSecondSegmentAnchorHitIndex];

        y2Vec[1] = hitsInGPU.lowEdgeYs[innerTripletSecondSegmentAnchorHitIndex];
        y2Vec[2] = hitsInGPU.highEdgeYs[innerTripletSecondSegmentAnchorHitIndex];

    }
    if(modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS)
    {
        x3Vec[1] = hitsInGPU.lowEdgeXs[innerTripletThirdSegmentAnchorHitIndex];
        x3Vec[2] = hitsInGPU.highEdgeXs[innerTripletThirdSegmentAnchorHitIndex];

        y3Vec[1] = hitsInGPU.lowEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
        y3Vec[2] = hitsInGPU.highEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
    }
    computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin2S, innerRadiusMax2S);

    for (int i=0; i<3; i++) {
      x1Vec[i] = x4;
      y1Vec[i] = y4;
    }
    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x1Vec[1] = hitsInGPU.lowEdgeXs[outerTripletSecondSegmentAnchorHitIndex];
        x1Vec[2] = hitsInGPU.highEdgeXs[outerTripletSecondSegmentAnchorHitIndex];

        y1Vec[1] = hitsInGPU.lowEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
        y1Vec[2] = hitsInGPU.highEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
    }
    computeErrorInRadius(x2Vec, y2Vec, x3Vec, y3Vec, x1Vec, y1Vec, bridgeRadiusMin2S, bridgeRadiusMax2S);

    for(int i=0; i<3; i++) {
      x2Vec[i] = x5;
      y2Vec[i] = y5;
    }
    if(modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS)
    {
        x2Vec[1] = hitsInGPU.lowEdgeXs[outerTripletThirdSegmentAnchorHitIndex];
        x2Vec[2] = hitsInGPU.highEdgeXs[outerTripletThirdSegmentAnchorHitIndex];

        y2Vec[1] = hitsInGPU.lowEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
        y2Vec[2] = hitsInGPU.highEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
    }
    computeErrorInRadius(x3Vec, y3Vec, x1Vec, y1Vec, x2Vec, y2Vec, outerRadiusMin2S, outerRadiusMax2S);

    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3);
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5);
    bridgeRadius = computeRadiusFromThreeAnchorHits(x2, y2, x3, y3, x4, y4);


    //computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin2S, innerRadiusMax2S);
    //computeErrorInRadius(x2Vec, y2Vec, x3Vec, y3Vec, x4Vec, y4Vec, bridgeRadiusMin2S, bridgeRadiusMax2S);
    //computeErrorInRadius(x3Vec, y3Vec, x4Vec, y4Vec, x5Vec, y5Vec, outerRadiusMin2S, outerRadiusMax2S);

    if(innerRadius < 0.95/(2 * k2Rinv1GeVf))
    {
        pass = false;
    }
    //split by category
    bool tempPass;
    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Barrel)
    {
       tempPass = matchRadiiBBBBB(innerRadius, bridgeRadius, outerRadius, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBBBE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        if(modulesInGPU.layers[lowerModuleIndex1] == 1)
        {
            tempPass = matchRadiiBBBEE12378(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
        }
        else if(modulesInGPU.layers[lowerModuleIndex1] == 2)
        {
            tempPass = matchRadiiBBBEE23478(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
        }
        else
        {
            tempPass = matchRadiiBBBEE34578(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
        }
    }

    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else
    {
        tempPass = matchRadiiEEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S,innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }

    pass = pass & tempPass;
    return pass;
}

__device__ bool SDL::checkIntervalOverlap(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */

__device__ bool SDL::matchRadiiBBBBB(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    float innerInvRadiusErrorBound =  0.1512;
    float bridgeInvRadiusErrorBound = 0.1781;
    float outerInvRadiusErrorBound = 0.1840;

    if(innerRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4449;
        bridgeInvRadiusErrorBound = 0.4033;
        outerInvRadiusErrorBound = 0.8016;
    }

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/bridgeRadiusMax, 1.0/bridgeRadiusMin);
}

__device__ bool SDL::matchRadiiBBBBE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1781;
    float bridgeInvRadiusErrorBound = 0.2167;
    float outerInvRadiusErrorBound = 1.1116;

    if(innerRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4750;
        bridgeInvRadiusErrorBound = 0.3903;
        outerInvRadiusErrorBound = 15.2120;
    }


    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f; //large number signifying infty

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/bridgeRadiusMax, 1.0/bridgeRadiusMin);
}

__device__ bool SDL::matchRadiiBBBEE12378(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    float innerInvRadiusErrorBound = 0.178;
    float bridgeInvRadiusErrorBound = 0.507;
    float outerInvRadiusErrorBound = 7.655;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S),1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));
}

__device__ bool SDL::matchRadiiBBBEE23478(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    float innerInvRadiusErrorBound = 0.2097;
    float bridgeInvRadiusErrorBound = 0.8557;
    float outerInvRadiusErrorBound = 24.0450;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE34578(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    float innerInvRadiusErrorBound = 0.066;
    float bridgeInvRadiusErrorBound = 0.617;
    float outerInvRadiusErrorBound = 2.688;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1840;
    float bridgeInvRadiusErrorBound = 0.5971;
    float outerInvRadiusErrorBound = 11.7102;

    if(innerRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as no selections
    {
        innerInvRadiusErrorBound = 1.0412;
        outerInvRadiusErrorBound = 32.2737;
        bridgeInvRadiusErrorBound = 10.9688;
    }

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{

    float innerInvRadiusErrorBound =  0.6376;
    float bridgeInvRadiusErrorBound = 2.1381;
    float outerInvRadiusErrorBound = 20.4179;

    if(innerRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as no selections!
    {
        innerInvRadiusErrorBound = 12.9173;
        outerInvRadiusErrorBound = 25.6702;
        bridgeInvRadiusErrorBound = 5.1700;
    }

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/innerRadiusMax, 1.0/innerRadiusMin, 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{

    float innerInvRadiusErrorBound =  1.9382;
    float bridgeInvRadiusErrorBound = 3.7280;
    float outerInvRadiusErrorBound = 5.7030;


    if(innerRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 23.2713;
        outerInvRadiusErrorBound = 24.0450;
        bridgeInvRadiusErrorBound = 21.7980;
    }

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/fmaxf(innerRadiusMax, innerRadiusMax2S), 1.0/fminf(innerRadiusMin, innerRadiusMin2S), 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiEEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    float innerInvRadiusErrorBound =  1.9382;
    float bridgeInvRadiusErrorBound = 2.2091;
    float outerInvRadiusErrorBound = 7.4084;

    if(innerRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 22.5226;
        bridgeInvRadiusErrorBound = 21.0966;
        outerInvRadiusErrorBound = 19.1252;
    }

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789.f;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789.f;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789.f;

    return checkIntervalOverlap(1.0/fmaxf(innerRadiusMax, innerRadiusMax2S), 1.0/fminf(innerRadiusMin, innerRadiusMin2S), 1.0/fmaxf(bridgeRadiusMax, bridgeRadiusMax2S), 1.0/fminf(bridgeRadiusMin, bridgeRadiusMin2S));

}

__device__ void SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& minimumRadius, float& maximumRadius)
{
    //brute force
    float candidateRadius;
    minimumRadius = 123456789.f;
    maximumRadius = 0;
    for(size_t i = 0; i < 3; i++)
    {
        float x1 = x1Vec[i];
	float y1 = y1Vec[i];
        for(size_t j = 0; j < 3; j++)
        {
	    float x2 = x2Vec[j];
	    float y2 = y2Vec[j];
            for(size_t k = 0; k < 3; k++)
            {
	       float x3 = x3Vec[k];
               float y3 = y3Vec[k];
               candidateRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3);
               maximumRadius = fmaxf(candidateRadius, maximumRadius);
               minimumRadius = fminf(candidateRadius, minimumRadius);
            }
        }
    }
}
__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.0/((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    float xy1sqr = x1 * x1 + y1 * y1;

    float xy2sqr = x2 * x2 + y2 * y2;

    float xy3sqr = x3 * x3 + y3 * y3;

    float g = 0.5 * ((y3 - y2) * xy1sqr + (y1 - y3) * xy2sqr + (y2 - y1) * xy3sqr) * denomInv;

    float f = 0.5 * ((x2 - x3) * xy1sqr + (x3 - x1) * xy2sqr + (x1 - x2) * xy3sqr) * denomInv;

    float c = ((x2 * y3 - x3 * y2) * xy1sqr + (x3 * y1 - x1 * y3) * xy2sqr + (x1 * y2 - x2 * y1) * xy3sqr) * denomInv;

    if(((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
	radius = -1;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index


    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}
