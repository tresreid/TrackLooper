#include "hip/hip_runtime.h"
# include "Event.cuh"
#include "allocate.h"


unsigned int N_MAX_HITS_PER_MODULE = 100;
const unsigned int N_MAX_MD_PER_MODULES = 100;
const unsigned int N_MAX_SEGMENTS_PER_MODULE = 600; //WHY!
const unsigned int MAX_CONNECTED_MODULES = 40;
const unsigned int N_MAX_TRACKLETS_PER_MODULE = 8000;//temporary
const unsigned int N_MAX_TRIPLETS_PER_MODULE = 5000;
const unsigned int N_MAX_TOTAL_TRIPLETS = 200000;
const unsigned int N_MAX_PIXEL_MD_PER_MODULES = 100000;
const unsigned int N_MAX_PIXEL_SEGMENTS_PER_MODULE = 50000;
const unsigned int N_MAX_QUINTUPLETS_PER_MODULE = 5000;
#ifdef FINAL_T3T4
const unsigned int N_MAX_TRACK_CANDIDATES_PER_MODULE = 50000;
const unsigned int N_MAX_PIXEL_TRACKLETS_PER_MODULE = 3000000;
const unsigned int N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE = 5000000;
#else
const unsigned int N_MAX_TRACK_CANDIDATES_PER_MODULE = 5000;
const unsigned int N_MAX_PIXEL_TRACKLETS_PER_MODULE = 200000;
const unsigned int N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE = 200000;
#endif
const unsigned int N_MAX_PIXEL_TRIPLETS = 3000000;

struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::pixelMap* SDL::pixelMapping = nullptr;
unsigned int SDL::nModules;

SDL::Event::Event()
{
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    trackletsInGPU = nullptr;
    pixelTrackletsInGPU = nullptr;
    tripletsInGPU = nullptr;
    quintupletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;
    pixelTripletsInGPU = nullptr;


    hitsInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    trackletsInCPU = nullptr;
    pixelTrackletsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    quintupletsInCPU = nullptr;
    pixelTripletsInCPU = nullptr;

    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_tracklets_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_tracklets_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    resetObjectsInModule();

}

SDL::Event::~Event()
{

#ifdef CACHE_ALLOC
    mdsInGPU->freeMemoryCache();
    segmentsInGPU->freeMemoryCache();
    tripletsInGPU->freeMemoryCache();
    pixelTrackletsInGPU->freeMemoryCache();
    trackCandidatesInGPU->freeMemoryCache();
#ifdef FINAL_T5
    quintupletsInGPU->freeMemoryCache();
#endif
#ifdef FINAL_T3T4
    trackletsInGPU->freeMemoryCache();
#endif
#else
    mdsInGPU->freeMemory();
    segmentsInGPU->freeMemory();
    tripletsInGPU->freeMemory();
    pixelTrackletsInGPU->freeMemory();
    trackCandidatesInGPU->freeMemory();
#ifdef FINAL_T5
    quintupletsInGPU->freeMemory();
#endif
#ifdef FINAL_T3T4
    trackletsInGPU->freeMemory();
#endif
#endif
    hipHostFree(mdsInGPU);
    hipHostFree(segmentsInGPU);
    hipHostFree(tripletsInGPU);
    hipHostFree(pixelTrackletsInGPU);
    hipHostFree(trackCandidatesInGPU);
    hitsInGPU->freeMemory();
    hipHostFree(hitsInGPU);

    pixelTripletsInGPU->freeMemory();
    hipHostFree(pixelTripletsInGPU);

#ifdef FINAL_T5
    hipHostFree(quintupletsInGPU);
#endif
#ifdef FINAL_T3T4
    hipHostFree(trackletsInGPU);
#endif

#ifdef Explicit_Hit
    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
#endif
#ifdef Explicit_MD
    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->hitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU;
    }
#endif
#ifdef Explicit_Seg
    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU;
    }
#endif
#ifdef Explicit_Tracklet
    if(trackletsInCPU != nullptr)
    {
        delete[] trackletsInCPU->segmentIndices;
        delete[] trackletsInCPU->nTracklets;
        delete[] trackletsInCPU->betaIn;
        delete[] trackletsInCPU->betaOut;
        delete[] trackletsInCPU->pt_beta;
        delete trackletsInCPU;
    }
    if(pixelTrackletsInCPU != nullptr)
    {
        delete[] pixelTrackletsInCPU->segmentIndices;
        delete pixelTrackletsInCPU->nPixelTracklets;
        delete[] pixelTrackletsInCPU->betaIn;
        delete[] pixelTrackletsInCPU->betaOut;
        delete[] pixelTrackletsInCPU->pt_beta;
        delete pixelTrackletsInCPU;
    }
#endif
#ifdef Explicit_Trips
    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete tripletsInCPU;
    }
#endif
#ifdef Explicit_T5
#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete quintupletsInCPU;
    }
#endif
#endif

#ifdef Explicit_PT3
    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU;
    }
#endif

#ifdef Explicit_Track
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete trackCandidatesInCPU;
    }
#endif
#ifdef Explicit_Module
    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->lowerModuleIndices;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->hitRanges;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->trackCandidateModuleIndices;
        delete[] modulesInCPU->quintupletModuleIndices;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;

        delete[] modulesInCPUFull->hitRanges;
        delete[] modulesInCPUFull->mdRanges;
        delete[] modulesInCPUFull->segmentRanges;
        delete[] modulesInCPUFull->trackletRanges;
        delete[] modulesInCPUFull->tripletRanges;
        delete[] modulesInCPUFull->trackCandidateRanges;

        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;

        delete[] modulesInCPUFull->lowerModuleIndices;
        delete[] modulesInCPUFull->reverseLookupLowerModuleIndices;
        delete[] modulesInCPUFull->trackCandidateModuleIndices;
        delete[] modulesInCPUFull->quintupletModuleIndices;
        delete[] modulesInCPUFull;
    }
#endif
}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    if(modulesInGPU == nullptr)
    {
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        //pixelMapping = new pixelMap;
        hipHostMalloc(&pixelMapping, sizeof(struct SDL::pixelMap));
        loadModulesFromFile(*modulesInGPU,nModules,*pixelMapping,moduleMetaDataFilePath); //nModules gets filled here
    }
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::cleanModules()
{
  #ifdef CACHE_ALLOC
  freeModulesCache(*modulesInGPU,*pixelMapping);
  #else
  freeModules(*modulesInGPU,*pixelMapping);
  #endif
  hipHostFree(modulesInGPU);
  hipHostFree(pixelMapping);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*modulesInGPU,nModules);
}

// Best working hit loading method. Previously named OMP
void SDL::Event::addHitToEvent(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    const int loopsize = x.size();// use the actual number of hits instead of a "max"

    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
    	  createHitsInExplicitMemory(*hitsInGPU, 2*loopsize); //unclear why but this has to be 2*loopsize to avoid crashing later (reported in tracklet allocation). seems to do with nHits values as well. this allows nhits to be set to the correct value of loopsize to get correct results without crashing. still beats the "max hits" so i think this is fine.
        #else
        createHitsInUnifiedMemory(*hitsInGPU,2*loopsize,0);
        #endif
    }


    float* host_x = &x[0]; // convert from std::vector to host array easily since vectors are ordered
    float* host_y = &y[0];
    float* host_z = &z[0];
    float* host_phis;
    float* host_etas;
    unsigned int* host_detId = &detId[0];
    unsigned int* host_idxs = &idxInNtuple[0];
    unsigned int* host_moduleIndex;
    float* host_rts;
    //float* host_idxs;
    float* host_highEdgeXs;
    float* host_highEdgeYs;
    float* host_lowEdgeXs;
    float* host_lowEdgeYs;
    hipHostMalloc(&host_moduleIndex,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_phis,sizeof(float)*loopsize);
    hipHostMalloc(&host_etas,sizeof(float)*loopsize);
    hipHostMalloc(&host_rts,sizeof(float)*loopsize);
    //hipHostMalloc(&host_idxs,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_highEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_highEdgeYs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeYs,sizeof(float)*loopsize);


    short* module_layers;
    short* module_subdet;
    int* module_hitRanges;
    ModuleType* module_moduleType;
    hipHostMalloc(&module_layers,sizeof(short)*nModules);
    hipHostMalloc(&module_subdet,sizeof(short)*nModules);
    hipHostMalloc(&module_hitRanges,sizeof(int)*2*nModules);
    hipHostMalloc(&module_moduleType,sizeof(ModuleType)*nModules);
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_subdet,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(module_moduleType,modulesInGPU->moduleType,nModules*sizeof(ModuleType),hipMemcpyDeviceToHost);


  for (int ihit=0; ihit<loopsize;ihit++){
    unsigned int moduleLayer = module_layers[(*detIdToIndex)[host_detId[ihit]]];
    unsigned int subdet = module_subdet[(*detIdToIndex)[host_detId[ihit]]];
    host_moduleIndex[ihit] = (*detIdToIndex)[host_detId[ihit]];


      host_rts[ihit] = sqrt(host_x[ihit]*host_x[ihit] + host_y[ihit]*host_y[ihit]);
      host_phis[ihit] = phi(host_x[ihit],host_y[ihit],host_z[ihit]);
      host_etas[ihit] = ((host_z[ihit]>0)-(host_z[ihit]<0))* std::acosh(sqrt(host_x[ihit]*host_x[ihit]+host_y[ihit]*host_y[ihit]+host_z[ihit]*host_z[ihit])/host_rts[ihit]);
//// This part i think has a race condition. so this is not run in parallel.
      unsigned int this_index = host_moduleIndex[ihit];
      if(module_subdet[this_index] == Endcap && module_moduleType[this_index] == TwoS)
      {
          float xhigh, yhigh, xlow, ylow;
          getEdgeHits(host_detId[ihit],host_x[ihit],host_y[ihit],xhigh,yhigh,xlow,ylow);
          host_highEdgeXs[ihit] = xhigh;
          host_highEdgeYs[ihit] = yhigh;
          host_lowEdgeXs[ihit] = xlow;
          host_lowEdgeYs[ihit] = ylow;

      }

      //set the hit ranges appropriately in the modules struct

      ////start the index rolling if the module is encountered for the first time
      ////always update the end index
      //modulesInGPU->hitRanges[this_index * 2 + 1] = ihit;
      //start the index rolling if the module is encountered for the first time
      if(module_hitRanges[this_index * 2] == -1)
      {
          module_hitRanges[this_index * 2] = ihit;
      }
      //always update the end index
      module_hitRanges[this_index * 2 + 1] = ihit;

  }
//simply copy the host arrays to the hitsInGPU struct
    hipMemcpy(hitsInGPU->xs,host_x,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->ys,host_y,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->zs,host_z,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->rts,host_rts,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->phis,host_phis,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->etas,host_etas,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->idxs,host_idxs,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->moduleIndices,host_moduleIndex,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeXs,host_highEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeYs,host_highEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeXs,host_lowEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeYs,host_lowEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->nHits,&loopsize,sizeof(unsigned int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipMemcpy(modulesInGPU->hitRanges,module_hitRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipDeviceSynchronize(); //doesn't seem to make a difference

    hipHostFree(host_rts);
    //hipHostFree(host_idxs);
    hipHostFree(host_phis);
    hipHostFree(host_etas);
    hipHostFree(host_moduleIndex);
    hipHostFree(host_highEdgeXs);
    hipHostFree(host_highEdgeYs);
    hipHostFree(host_lowEdgeXs);
    hipHostFree(host_lowEdgeYs);
    hipHostFree(module_layers);
    hipHostFree(module_subdet);
    hipHostFree(module_hitRanges);
    hipHostFree(module_moduleType);

}
__global__ void addPixelSegmentToEventKernel(unsigned int* hitIndices0,unsigned int* hitIndices1,unsigned int* hitIndices2,unsigned int* hitIndices3, float* dPhiChange, float* ptIn, float* ptErr, float* px, float* py, float* pz, float* eta, float* etaErr,float* phi, unsigned int pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU,const int size, int* superbin, int* pixelType)
{

    for( int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x*gridDim.x)
    {

      unsigned int innerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid);
      unsigned int outerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid) +1;
      unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + tid;
#ifdef DUP_RM
    bool dup = false;
    for (int i=0; i<tid; i++){
        if(abs(eta[i] - eta[tid]) > 0.3){continue;}
        if(abs(phi[i] - phi[tid]) > 0.3){continue;}
        //if(abs(ptIn[i] - ptIn[tid])/ptIn[tid] > 0.3){continue;}
        float dR2 = (eta[i]-eta[tid])*(eta[i]-eta[tid]) + (phi[i]-phi[tid])*(phi[i]-phi[tid]);
        //if(dR2 > 0.001){continue;}
        dup = true;
        break;
      }
      if(!dup){
#ifdef CUT_VALUE_DEBUG
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif
      addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], eta[tid], phi[tid], pixelSegmentIndex, tid, superbin[tid], pixelType[tid]);
    }
#else
#ifdef CUT_VALUE_DEBUG
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif
      addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], eta[tid], phi[tid], pixelSegmentIndex, tid, superbin[tid], pixelType[tid]);
#endif
    }
}
void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> eta, std::vector<float> etaErr, std::vector<float> phi, std::vector<int> superbin, std::vector<int> pixelType)
{
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    const int size = ptIn.size();
    unsigned int pixelModuleIndex = (*detIdToIndex)[1];
    unsigned int* hitIndices0_host = &hitIndices0[0];
    unsigned int* hitIndices1_host = &hitIndices1[0];
    unsigned int* hitIndices2_host = &hitIndices2[0];
    unsigned int* hitIndices3_host = &hitIndices3[0];
    float* dPhiChange_host = &dPhiChange[0];
    float* ptIn_host = &ptIn[0];
    float* ptErr_host = &ptErr[0];
    float* px_host = &px[0];
    float* py_host = &py[0];
    float* pz_host = &pz[0];
    float* etaErr_host = &etaErr[0];
    float* eta_host = &eta[0];
    float* phi_host = &phi[0];
    int* superbin_host = &superbin[0];
    int* pixelType_host = &pixelType[0];

    unsigned int* hitIndices0_dev;
    unsigned int* hitIndices1_dev;
    unsigned int* hitIndices2_dev;
    unsigned int* hitIndices3_dev;
    float* dPhiChange_dev;
    float* ptIn_dev;
    float* ptErr_dev;
    float* px_dev;
    float* py_dev;
    float* pz_dev;
    float* etaErr_dev;
    float* eta_dev;
    float* phi_dev;
    int* superbin_dev;
    int* pixelType_dev;

    hipMalloc(&hitIndices0_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices1_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices2_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices3_dev,size*sizeof(unsigned int));
    hipMalloc(&dPhiChange_dev,size*sizeof(unsigned int));
    hipMalloc(&ptIn_dev,size*sizeof(unsigned int));
    hipMalloc(&ptErr_dev,size*sizeof(unsigned int));
    hipMalloc(&px_dev,size*sizeof(unsigned int));
    hipMalloc(&py_dev,size*sizeof(unsigned int));
    hipMalloc(&pz_dev,size*sizeof(unsigned int));
    hipMalloc(&etaErr_dev,size*sizeof(unsigned int));
    hipMalloc(&eta_dev, size*sizeof(unsigned int));
    hipMalloc(&phi_dev, size*sizeof(unsigned int));
    hipMalloc(&superbin_dev,size*sizeof(int));
    hipMalloc(&pixelType_dev,size*sizeof(int));

    hipMemcpy(hitIndices0_dev,hitIndices0_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices1_dev,hitIndices1_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices2_dev,hitIndices2_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices3_dev,hitIndices3_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(dPhiChange_dev,dPhiChange_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptIn_dev,ptIn_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptErr_dev,ptErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(px_dev,px_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(py_dev,py_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(pz_dev,pz_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(etaErr_dev,etaErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(eta_dev, eta_host, size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(phi_dev, phi_host, size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(superbin_dev,superbin_host,size*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(pixelType_dev,pixelType_host,size*sizeof(int),hipMemcpyHostToDevice);

    unsigned int nThreads = 256;
    unsigned int nBlocks =  size % nThreads == 0 ? size/nThreads : size/nThreads + 1;

  addPixelSegmentToEventKernel<<<nBlocks,nThreads>>>(hitIndices0_dev,hitIndices1_dev,hitIndices2_dev,hitIndices3_dev,dPhiChange_dev,ptIn_dev,ptErr_dev,px_dev,py_dev,pz_dev,eta_dev, etaErr_dev, phi_dev, pixelModuleIndex, *modulesInGPU,*hitsInGPU,*mdsInGPU,*segmentsInGPU,size, superbin_dev, pixelType_dev);
   //std::cout<<"Number of pixel segments = "<<size<<std::endl;
   hipDeviceSynchronize();
   hipMemcpy(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice);
   unsigned int mdSize = 2 * size;
   hipMemcpy(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice);

  hipFree(hitIndices0_dev);
  hipFree(hitIndices1_dev);
  hipFree(hitIndices2_dev);
  hipFree(hitIndices3_dev);
  hipFree(dPhiChange_dev);
  hipFree(ptIn_dev);
  hipFree(ptErr_dev);
  hipFree(px_dev);
  hipFree(py_dev);
  hipFree(pz_dev);
  hipFree(etaErr_dev);
  hipFree(eta_dev);
  hipFree(phi_dev);
  hipFree(superbin_dev);
  hipFree(pixelType_dev);
}

void SDL::Event::addMiniDoubletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(mdsInGPU->nMDs[idx] == 0 or modulesInGPU->hitRanges[idx * 2] == -1)
        {
            modulesInGPU->mdRanges[idx * 2] = -1;
            modulesInGPU->mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            modulesInGPU->mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + mdsInGPU->nMDs[idx] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[idx] -1] += mdsInGPU->nMDs[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += mdsInGPU->nMDs[idx];
            }

        }
    }
}
void SDL::Event::addMiniDoubletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
unsigned int* nMDsCPU;
hipHostMalloc(&nMDsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nMDsCPU,mdsInGPU->nMDs,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_mdRanges;
hipHostMalloc(&module_mdRanges, nModules* 2*sizeof(int));
hipMemcpy(module_mdRanges,modulesInGPU->mdRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
int* module_hitRanges;
hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nMDsCPU[idx] == 0 or module_hitRanges[idx * 2] == -1)
        {
            module_mdRanges[idx * 2] = -1;
            module_mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            module_mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + nMDsCPU[idx] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[idx] -1] += nMDsCPU[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[idx] - 1] += nMDsCPU[idx];
            }

        }
    }
hipMemcpy(modulesInGPU->mdRanges,module_mdRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);
hipHostFree(nMDsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_mdRanges);
hipHostFree(module_layers);
hipHostFree(module_hitRanges);
}

void SDL::Event::addSegmentsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(segmentsInGPU->nSegments[idx] == 0)
        {
            modulesInGPU->segmentRanges[idx * 2] = -1;
            modulesInGPU->segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            modulesInGPU->segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU->nSegments[idx] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += segmentsInGPU->nSegments[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[idx] -1] += segmentsInGPU->nSegments[idx];
            }
        }
    }
}
void SDL::Event::addSegmentsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nSegmentsCPU;
hipHostMalloc(&nSegmentsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nSegmentsCPU,segmentsInGPU->nSegments,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_segmentRanges;
hipHostMalloc(&module_segmentRanges, nModules* 2*sizeof(int));
hipMemcpy(module_segmentRanges,modulesInGPU->segmentRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nSegmentsCPU[idx] == 0)
        {
            module_segmentRanges[idx * 2] = -1;
            module_segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            module_segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + nSegmentsCPU[idx] - 1;

            if(module_subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[module_layers[idx] - 1] += nSegmentsCPU[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[idx] -1] += nSegmentsCPU[idx];
            }
        }
    }
hipHostFree(nSegmentsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_segmentRanges);
hipHostFree(module_layers);
}

void SDL::Event::createMiniDoublets()
{
    hipDeviceSynchronize();
    auto memStart = std::chrono::high_resolution_clock::now();
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
        //FIXME: Add memory locations for pixel MDs
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    hipDeviceSynchronize();
    auto memStop = std::chrono::high_resolution_clock::now();
    auto memDuration = std::chrono::duration_cast<std::chrono::milliseconds>(memStop - memStart); //in milliseconds

    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    int nThreads = 1;
    int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_MD
    int maxThreadsPerModule=0;
    #ifdef Explicit_Module
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_hitRanges;
    hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    bool* module_isLower;
    hipHostMalloc(&module_isLower, nModules*sizeof(bool));
    hipMemcpy(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost);
    bool* module_isInverted;
    hipHostMalloc(&module_isInverted, nModules*sizeof(bool));
    hipMemcpy(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = module_lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndexExplicit(lowerModuleIndex,module_isLower[lowerModuleIndex],module_isInverted[lowerModuleIndex]);
      int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
      int upperHitRanges = module_hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_hitRanges);
    hipHostFree(module_isLower);
    hipHostFree(module_isInverted);
    #else
    //int maxThreadsPerModule=0;
    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndex(lowerModuleIndex);
      int lowerHitRanges = modulesInGPU->hitRanges[lowerModuleIndex*2];
      int upperHitRanges = modulesInGPU->hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = modulesInGPU->hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = modulesInGPU->hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    #endif
    //printf("maxThreadsPerModule=%d\n", maxThreadsPerModule);
    dim3 nThreads(1,128);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1), (maxThreadsPerModule % nThreads.y == 0 ? maxThreadsPerModule/nThreads.y : maxThreadsPerModule/nThreads.y + 1));
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1),(N_MAX_HITS_PER_MODULE % nThreads.y == 0 ? N_MAX_HITS_PER_MODULE/nThreads.y : N_MAX_HITS_PER_MODULE/nThreads.y + 1), (N_MAX_HITS_PER_MODULE % nThreads.z == 0 ? N_MAX_HITS_PER_MODULE/nThreads.z : N_MAX_HITS_PER_MODULE/nThreads.z + 1));
#endif
#endif

    hipDeviceSynchronize();
    auto syncStart = std::chrono::high_resolution_clock::now();

    createMiniDoubletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    auto syncStop = std::chrono::high_resolution_clock::now();

    auto syncDuration =  std::chrono::duration_cast<std::chrono::milliseconds>(syncStop - syncStart);

    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

#if defined(AddObjects)
#ifdef Explicit_MD
    addMiniDoubletsToEventExplicit();
#else
    addMiniDoubletsToEvent();
#endif
#endif


}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_Seg
    int max_cModules=0;
    int sq_max_nMDs = 0;
    int nonZeroModules = 0;
  #ifdef Explicit_Module
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_moduleMap;
    hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
    hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = module_moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
    hipHostFree(module_moduleMap);
  #else

    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = modulesInGPU->moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
  #endif
    //printf("max nConnectedModules=%d nonZeroModules=%d max sq_max_nMDs=%d\n", max_cModules, nonZeroModules, sq_max_nMDs);
    dim3 nThreads(256,1,1);
    dim3 nBlocks((sq_max_nMDs%nThreads.x==0 ? sq_max_nMDs/nThreads.x : sq_max_nMDs/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nLowerModules%nThreads.z==0 ? nLowerModules/nThreads.z : nLowerModules/nThreads.z + 1));
    free(nMDs);
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks(((nLowerModules * MAX_CONNECTED_MODULES)  % nThreads.x == 0 ? (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x : (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x + 1),(N_MAX_MD_PER_MODULES % nThreads.y == 0 ? N_MAX_MD_PER_MODULES/nThreads.y : N_MAX_MD_PER_MODULES/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0  ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));
#endif
#endif

    createSegmentsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#if defined(AddObjects)
#ifdef Explicit_Seg
    addSegmentsToEventExplicit();
#else
    addSegmentsToEvent();
#endif
#endif

}


void SDL::Event::createTriplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    if(tripletsInGPU == nullptr)
    {
        hipHostMalloc(&tripletsInGPU, sizeof(SDL::triplets));
#ifdef Explicit_Trips
        createTripletsInExplicitMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#else
        createTripletsInUnifiedMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Trips
  #ifdef Explicit_Module
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    //unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
  #else
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
  #endif
    hipMemcpy(index_gpu, index, nonZeroModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    int max_OuterSeg = 0;
    max_OuterSeg = N_MAX_SEGMENTS_PER_MODULE;
    dim3 nThreads(16,16,1);
    dim3 nBlocks((max_OuterSeg % nThreads.x == 0 ? max_OuterSeg / nThreads.x : max_OuterSeg / nThreads.x + 1),(max_InnerSeg % nThreads.y == 0 ? max_InnerSeg/nThreads.y : max_InnerSeg/nThreads.y + 1), (nonZeroModules % nThreads.z == 0 ? nonZeroModules/nThreads.z : nonZeroModules/nThreads.z + 1));
    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, index_gpu);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    free(nSegments);
    free(index);
    hipFree(index_gpu);
#else
    printf("original 3D grid launching in createTriplets does not exist");
    exit(1);
#endif
#endif

#if defined(AddObjects)
#ifdef Explicit_Trips
    addTripletsToEventExplicit();
#else
    addTripletsToEvent();
#endif
#endif
}

void SDL::Event::createTrackletsWithModuleMap()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        //FIXME:Add memory locations for pixel tracklets
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE, nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    #ifdef T4FromT3
      createTrackletsFromTriplets<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *trackletsInGPU);
    #else
      createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);
    #endif

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Tracklet
  #ifdef T4FromT3
    int threadSize=230000;
    unsigned int *nTriplets = (unsigned int*)malloc((nLowerModules-1)*sizeof(unsigned int));
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, (nLowerModules-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int totalCand=0;
    for (int i=0; i< nLowerModules-1; i++) {
      unsigned int nInnerTriplets = nTriplets[i];
      if(nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      if (nInnerTriplets !=0) {
        for (int k=0; k<nInnerTriplets; k++) {
          threadIdx[totalCand+k] = i;
          threadIdx_offset[totalCand+k] = k;
        }
        totalCand += nInnerTriplets;
      }
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 32, 1);
    dim3 nBlocks((N_MAX_TRIPLETS_PER_MODULE % nThreads.x == 0 ? N_MAX_TRIPLETS_PER_MODULE/nThreads.x : N_MAX_TRIPLETS_PER_MODULE/nThreads.x + 1), (totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);

    createTrackletsFromTriplets<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *trackletsInGPU,threadIdx_gpu,threadIdx_gpu_offset);
    free(threadIdx);
    hipFree(threadIdx_gpu);
    free(nTriplets);

  #else
      int max_cModules = 0;
      int sq_max_segments = 0;
      int nonZeroSegModules = 0;
      int inner_max_segments = 0;
      int outer_max_segments = 0;
      unsigned int *index_gpu;
      unsigned int *outerLowerModuleIndices = (unsigned int*)malloc(nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int));
      unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
      unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
      hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    #ifdef Explicit_Module
      hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
      unsigned int* module_lowerModuleIndices;
      hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
      hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
      unsigned int* module_nConnectedModules;
      hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
      hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
      unsigned int* module_moduleMap;
      hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
      hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);
      for (int i=0; i<nLowerModules; i++) {
        unsigned int innerInnerLowerModuleIndex = module_lowerModuleIndices[i];
        unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex];
        if (nInnerSegments!=0) {
          index[nonZeroSegModules] = i;
          nonZeroSegModules++;
        }
        inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

        for (int j=0; j<nInnerSegments; j++) {
          unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
          unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
          unsigned int nOuterInnerLowerModules = module_nConnectedModules[innerOuterLowerModuleIndex];
          max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
          for (int k=0; k<nOuterInnerLowerModules; k++) {
            unsigned int outerInnerLowerModuleIndex = module_moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
            unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex];
            sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
          }
        }
      }
      hipHostFree(module_lowerModuleIndices);
      hipHostFree(module_nConnectedModules);
      hipHostFree(module_moduleMap);
    #else
      //unsigned int nModules = *modulesInGPU->nModules;
      hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
      for (int i=0; i<nLowerModules; i++) {
        unsigned int innerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
        unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE  ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerInnerLowerModuleIndex];
        if (nInnerSegments!=0) {
          index[nonZeroSegModules] = i;
          nonZeroSegModules++;
        }
        inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

        for (int j=0; j<nInnerSegments; j++) {
          unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
          unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
          unsigned int nOuterInnerLowerModules = modulesInGPU->nConnectedModules[innerOuterLowerModuleIndex];
          max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
          for (int k=0; k<nOuterInnerLowerModules; k++) {
            unsigned int outerInnerLowerModuleIndex = modulesInGPU->moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
            unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
            sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
          }
        }
      }
    #endif
    hipMemcpy(index_gpu, index, nonZeroSegModules*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(128,1,1);
    dim3 nBlocks((sq_max_segments%nThreads.x==0 ? sq_max_segments/nThreads.x : sq_max_segments/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nonZeroSegModules%nThreads.z==0 ? nonZeroSegModules/nThreads.z : nonZeroSegModules/nThreads.z + 1));

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, index_gpu);
    free(outerLowerModuleIndices);
    free(nSegments);
    free(index);
    hipFree(index_gpu);
  #endif
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }


#else
    printf("original 3D grid launching in createTracklets does not exist");
    exit(1);
#endif
#endif
#if defined(AddObjects)
#ifdef Explicit_Tracklet
    addTrackletsToEventExplicit();
#else
    addTrackletsToEvent();
#endif
#endif

}

void SDL::Event::createPixelTrackletsWithMap()
{
    if(pixelTrackletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTrackletsInGPU, sizeof(SDL::pixelTracklets));
#ifdef Explicit_Tracklet
        createPixelTrackletsInExplicitMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#else
        createPixelTrackletsInUnifiedMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#endif
    }
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_Pixel
    unsigned int pixelModuleIndex;
    unsigned int nInnerSegments;
    int* superbins;
    int* pixelTypes;
#ifdef Explicit_Module
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    pixelModuleIndex = nModules-1;
    unsigned int* nSegments;
    hipHostMalloc(& nSegments,nModules*sizeof(unsigned int));
    hipMemcpy(nSegments,segmentsInGPU->nSegments,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex]; // number of pLS
    hipHostMalloc(& superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(& pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipMemcpy(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);

#else
    pixelModuleIndex = *modulesInGPU->nModules - 1; // pixel module index
    nInnerSegments = segmentsInGPU->nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU->nSegments[pixelModuleIndex]; // number of pLS
    superbins = segmentsInGPU->superbin;
    pixelTypes = segmentsInGPU->pixelType;
#endif
////////DUPs
float* etas;
float* phis;
float* pts;
    hipHostMalloc(&etas,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float));
    hipHostMalloc(&phis,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float));
    hipHostMalloc(&pts,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float));
    hipMemcpy(etas,segmentsInGPU->eta,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(phis,segmentsInGPU->phi,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(pts,segmentsInGPU->ptIn,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(float),hipMemcpyDeviceToHost);

/////////
    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));
    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemset(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int)); // so if not set, it will pass in the kernel
    unsigned int totalSegs=0;
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelIndexOffsetPos;
    int i =-1;
    for (int ix=0; ix < nInnerSegments;ix++){// loop over # pLS
      int pixelType = pixelTypes[ix];// get pixel type for this pLS
      int superbin = superbins[ix]; //get superbin for this pixel
      if(superbin <0) {/*printf("bad neg %d\n",ix);*/continue;}
      if(superbin >=45000) {/*printf("bad pos %d %d %d\n",ix,superbin,pixelType);*/continue;}// skip any weird out of range values
      if(pixelType >2 || pixelType < 0){/*printf("bad pixel type %d %d\n",ix,pixelType);*/continue;}
/////////////////DUP
//    bool dup = false;
//    for (int jx=0; jx<ix; jx++){
//        float dEta = abs(etas[ix] - etas[jx]);
//        float dPhi = abs(phis[ix] - phis[jx]);
//        if(dPhi > M_PI){dPhi= dPhi - 2*M_PI;}
//        if( dEta> 0.0003){continue;}
//        if( dPhi > 0.0003){continue;}
//        //if(abs(pts[ix] - pts[jx])/pts[jx] > 0.3){continue;}
//        float dR2 = dEta*dEta + dPhi*dPhi; 
//        if(dR2 > 0.000000000){continue;}
//        //printf("dR2: %e %f %f \n",dR2,pts[ix],pts[jx]);
//        dup = true;
//        break;
//      }
//      if(dup){continue;}
///////////////////////
      i++;
      if(pixelType ==0){ // used pixel type to select correct size-index arrays
        connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
        connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
        for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++){ // loop over modules from the size
          segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
          segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
        }
        totalSegs += connectedPixelSize_host[i]; // increment counter
      if (pixelMapping->connectedPixelsSizes[superbin] > max_size){ max_size = pixelMapping->connectedPixelsSizes[superbin];} // set the maximum number of modules in a row
      }
      else if(pixelType ==1){
        connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
        connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
        for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++){
          segs_pix[totalSegs+j] = i;
          segs_pix_offset[totalSegs+j] = j;
        }
        totalSegs += connectedPixelSize_host[i];
      if (pixelMapping->connectedPixelsSizesPos[superbin]> max_size){ max_size = pixelMapping->connectedPixelsSizesPos[superbin];}
      }
      else if(pixelType ==2){
        connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
        connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
        for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++){
          segs_pix[totalSegs+j] = i;
          segs_pix_offset[totalSegs+j] = j;
        }
        totalSegs += connectedPixelSize_host[i];
      if (pixelMapping->connectedPixelsSizesNeg[superbin] > max_size){max_size = pixelMapping->connectedPixelsSizesNeg[superbin];}
      }
      else{continue;}
    }

    hipMemcpy(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(32,16,1);
    dim3 nBlocks((totalSegs % nThreads.x == 0 ? totalSegs / nThreads.x : totalSegs / nThreads.x + 1),
                  (max_size % nThreads.y == 0 ? max_size/nThreads.y : max_size/nThreads.y + 1),1);
    createPixelTrackletsInGPUFromMap<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU,
    connectedPixelSize_dev,connectedPixelIndex_dev,i,segs_pix_gpu,segs_pix_gpu_offset);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

      }
    //dim3 nThreads_dup(32,32,1);
    //dim3 nBlocks_dup(16,16,1);
    dim3 nThreads_dup(1024,1,1); //about exhaustive
    dim3 nBlocks_dup(64,1,1);
    removeDupPixelTrackletsInGPUFromMap<<<nThreads_dup,nBlocks_dup>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU);
    //removeDupPixelTrackletsInGPUFromMap<<<1,1>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU,
    hipDeviceSynchronize();

    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
#ifdef Explicit_Module
    hipHostFree(nSegments);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
#endif
    free(segs_pix);
    hipFree(segs_pix_gpu);

#else
    printf("original 3D grid launching in createPixelTracklets does not exist");
    exit(2);
#endif
#endif

    unsigned int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, &(pixelTrackletsInGPU->nPixelTracklets), sizeof(unsigned int), hipMemcpyDeviceToHost);
}

void SDL::Event::createPixelTracklets()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    if(pixelTrackletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTrackletsInGPU, sizeof(SDL::pixelTracklets));
#ifdef Explicit_Tracklet
        createPixelTrackletsInExplicitMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#else
        createPixelTrackletsInUnifiedMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_Pixel
#ifdef Explicit_Module
    unsigned int nModules; //= *modulesInGPU->nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
#else
    unsigned int nModules = *modulesInGPU->nModules;
#endif
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int pixelModuleIndex = nModules - 1;
    unsigned int nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex];
#ifdef Explicit_Module
    unsigned int* lowerModuleIndices;
    hipHostMalloc(&lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
#endif
    int threadSize = 100000;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    unsigned int totalCand=0;
    for (int i=0; i<nLowerModules; i++) {
#ifdef Explicit_Module
      unsigned int outerInnerLowerModuleIndex = lowerModuleIndices[i];
#else
      unsigned int outerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
#endif
      unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
      if (nOuterSegments!=0) {
	for (int k=0; k<nOuterSegments; k++) {
          threadIdx[totalCand+k] = i;
          threadIdx_offset[totalCand+k] = k;
        }
	totalCand += nOuterSegments;
      }
    }

    if (threadSize < totalCand) {
      printf("threadSize=%d totalCand=%d: increase buffer size for threadIdx in createPixelTracklets\n", threadSize, totalCand);
      exit(1);
    }

    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16,32,1);
    dim3 nBlocks((nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1),(totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);
    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, threadIdx_gpu, threadIdx_gpu_offset);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
    	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

    free(nSegments);
    free(threadIdx);
    hipFree(threadIdx_gpu);
#ifdef Explicit_Module
    hipHostFree(lowerModuleIndices);
#endif

#else
    printf("original 3D grid launching in createPixelTracklets does not exist");
    exit(2);
#endif
#endif

    unsigned int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, &(pixelTrackletsInGPU->nPixelTracklets), sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef Warnings
    std::cout<<"number of pixel tracklets = "<<nPixelTracklets<<std::endl;
#endif
}

void SDL::Event::createTrackCandidates()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules + 1; //including the pixel module
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    nLowerModules += 1;// include the pixel module

    //construct the list of eligible modules
    unsigned int nEligibleModules = 0;
    createEligibleModulesListForTrackCandidates(*modulesInGPU, nEligibleModules, N_MAX_TRACK_CANDIDATES_PER_MODULE);

    if(trackCandidatesInGPU == nullptr)
    {
        hipHostMalloc(&trackCandidatesInGPU, sizeof(SDL::trackCandidates));
#ifdef Explicit_Track
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#else
        createTrackCandidatesInUnifiedMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#endif
    }

#ifdef FINAL_pT2
    printf("running final state pT2\n");
    unsigned int nThreadsx = 1;
    unsigned int nBlocksx = ( N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE) % nThreadsx == 0 ? N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE/nThreadsx : N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE/nThreadsx + 1;
    addpT2asTrackCandidateInGPU<<<nBlocksx,nThreadsx>>>(*modulesInGPU,*pixelTrackletsInGPU,*trackCandidatesInGPU);
    hipError_t cudaerr_pT2 = hipDeviceSynchronize();
    if(cudaerr_pT2 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT2)<<std::endl;
    }
#elif FINAL_pT3
    printf("running final state pT3\n");
    unsigned int nThreadsx = 1;
    unsigned int nBlocksx = (N_MAX_PIXEL_TRIPLETS) % nThreadsx == 0 ? N_MAX_PIXEL_TRIPLETS / nThreadsx : N_MAX_PIXEL_TRIPLETS / nThreadsx + 1;
    addpT3asTrackCandidateInGPU<<<nBlocksx, nThreadsx>>>(*modulesInGPU, *pixelTripletsInGPU, *trackCandidatesInGPU);
    hipError_t cudaerr_pT3 = hipDeviceSynchronize();
    if(cudaerr_pT3 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT3)<<std::endl;
    }
#endif // final state pT2 and pT3
#ifdef FINAL_T5
    printf("running final state T5\n");
    dim3 nThreads(32,16,1);
    dim3 nBlocks(((nLowerModules-1) % nThreads.x == 0 ? (nLowerModules-1)/nThreads.x : (nLowerModules-1)/nThreads.x + 1),((N_MAX_QUINTUPLETS_PER_MODULE-1) % nThreads.y == 0 ? (N_MAX_QUINTUPLETS_PER_MODULE-1)/nThreads.y : (N_MAX_QUINTUPLETS_PER_MODULE-1)/nThreads.y + 1),1);
    addT5asTrackCandidateInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*quintupletsInGPU,*trackCandidatesInGPU);

    hipError_t cudaerr_T5 = hipDeviceSynchronize();
    if(cudaerr_T5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_T5)<<std::endl;
    }
#endif // final state T5



#ifdef FINAL_T3T4
    printf("running final state T3T4\n");
#ifdef NESTED_PARA
    //auto t0 = std::chrono::high_resolution_clock::now();
    unsigned int nThreads = 1;
    unsigned int nBlocks = (nLowerModules-1) % nThreads == 0 ? (nLowerModules-1)/nThreads : (nLowerModules-1)/nThreads + 1;

    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

    //Pixel Track Candidates created separately
    nThreads = 1;
    nBlocks = (nLowerModules - 1) % nThreads == 0 ? (nLowerModules - 1)/nThreads : (nLowerModules - 1)/nThreads + 1;

    createPixelTrackCandidatesInGPU<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU);

    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
#ifdef NEWGRID_Track
    //auto t0 = std::chrono::high_resolution_clock::now();
    int maxOuterTr = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);
    unsigned int *nTriplets = (unsigned int*)malloc((2*nLowerModules-1)*sizeof(unsigned int));
    //unsigned int *nTracklets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *nTracklets = nTriplets + nLowerModules -1;
    //int threadSize=2300000;
    int threadSize=10000000;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, (nLowerModules-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(nTracklets, trackletsInGPU->nTracklets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    int nPixelTracklets;    
    hipMemcpy(&nPixelTracklets, pixelTrackletsInGPU->nPixelTracklets, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
      nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;
    unsigned int totalCand=0;
    for (int i=0; i< nLowerModules-1; i++) {
      unsigned int nInnerTracklets = nTracklets[i];
      if(nInnerTracklets > N_MAX_TRACKLETS_PER_MODULE)
	nInnerTracklets = N_MAX_TRACKLETS_PER_MODULE;
      unsigned int nInnerTriplets = nTriplets[i];
      if(nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      unsigned int temp = max(nInnerTracklets, nInnerTriplets);
      if (temp !=0) {
        for (int k=0; k<temp; k++) {
          threadIdx[totalCand+k] = i;
          //printf("totalCand+k: %d\n",totalCand+k);
          threadIdx_offset[totalCand+k] = k;
        }
	totalCand += temp;
      }
    }
    if (threadSize < totalCand) {
      printf("threadSize=%d totalCand=%d: Increase buffer size for threadIdx in createTrackCandidates\n", threadSize, totalCand);
      exit(2);
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 32, 1);
    dim3 nBlocks((maxOuterTr % nThreads.x == 0 ? maxOuterTr/nThreads.x : maxOuterTr/nThreads.x + 1), (totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);
    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    dim3 nThreads_p(16,16,1);
    dim3 nBlocks_p((nPixelTracklets % nThreads_p.x == 0 ? nPixelTracklets/nThreads_p.x : nPixelTracklets/nThreads_p.x + 1), (totalCand % nThreads_p.y == 0 ? totalCand/nThreads_p.y : totalCand/nThreads_p.y + 1), 1);
    createPixelTrackCandidatesInGPU<<<nBlocks_p, nThreads_p>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }

    free(threadIdx);
    free(nTriplets);
    hipFree(threadIdx_gpu);
#else
    printf("original 3D grid launching in createTrackCandidates does not exist");
    exit(3);
#endif
#endif 
#endif // Final state T3+T4
    dim3 nThreads_dup(32, 32, 1);
    dim3 nBlocks_dup(16, 16, 1);
  removeDupTrackCandidates<<<nBlocks_dup,nThreads_dup>>>(*trackCandidatesInGPU,*modulesInGPU);
  //removeDupTrackCandidates<<<1,1>>>(*trackCandidatesInGPU,*modulesInGPU);
  hipDeviceSynchronize();
#if defined(AddObjects)
#ifdef Explicit_Track
    addTrackCandidatesToEventExplicit();
#else
    addTrackCandidatesToEvent();
#endif
#endif

}

void SDL::Event::createPixelTriplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    if(pixelTripletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTripletsInGPU, sizeof(SDL::pixelTriplets));
    }
#ifdef Explicit_PT3
    createPixelTripletsInExplicitMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS);
#else
    createPixelTripletsInUnifiedMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS);
#endif

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules / nThreads : nLowerModules / nThreads + 1;

    createPixelTripletsInGPU<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }

    unsigned int nPixelTriplets;
    hipMemcpy(&nPixelTriplets, &(pixelTripletsInGPU->nPixelTriplets),  sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef Warnings
    std::cout<<"number of pixel triplets = "<<nPixelTriplets<<std::endl;
#endif

}


void SDL::Event::createQuintuplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int nEligibleT5Modules = 0;
    unsigned int *indicesOfEligibleModules = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));

    unsigned int maxTriplets;
    createEligibleModulesListForQuintuplets(*modulesInGPU, *tripletsInGPU, nEligibleT5Modules, indicesOfEligibleModules, N_MAX_QUINTUPLETS_PER_MODULE, maxTriplets);

    if(quintupletsInGPU == nullptr)
    {
        hipHostMalloc(&quintupletsInGPU, sizeof(SDL::quintuplets));
#ifdef Explicit_T5
        createQuintupletsInExplicitMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules);
#else
        createQuintupletsInUnifiedMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules);
#endif
    }


#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
    createQuintupletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU);


    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
#ifdef NEWGRID_T5
    int threadSize=N_MAX_TOTAL_TRIPLETS;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));

    unsigned int *nTriplets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost);

    int nTotalTriplets = 0;
    for (int i=0; i<nEligibleT5Modules; i++) {
      int index = indicesOfEligibleModules[i];
      unsigned int nInnerTriplets = nTriplets[index];
      if (nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE) nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      if (nInnerTriplets !=0) {
        for (int j=0; j<nInnerTriplets; j++) {
          threadIdx[nTotalTriplets + j] = index;
          threadIdx_offset[nTotalTriplets + j] = j;
        }
        nTotalTriplets += nInnerTriplets;
      }
    }
    printf("T5: nTotalTriplets=%d nEligibleT5Modules=%d\n", nTotalTriplets, nEligibleT5Modules);
    if (threadSize < nTotalTriplets) {
      printf("threadSize=%d nTotalTriplets=%d: Increase buffer size for threadIdx in createQuintuplets\n", threadSize, nTotalTriplets);
      exit(1);
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 16, 1);
    int max_outerTriplets = N_MAX_TRIPLETS_PER_MODULE;
    dim3 nBlocks((max_outerTriplets % nThreads.x == 0 ? max_outerTriplets/nThreads.x : max_outerTriplets/nThreads.x + 1), (nTotalTriplets % nThreads.y == 0 ? nTotalTriplets/nThreads.y : nTotalTriplets/nThreads.y + 1), 1);
    createQuintupletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    //dim3 dupThreads(1,1,1);
    //dim3 dupBlocks(1,1,1);
    dim3 dupThreads(32,32,1);
    dim3 dupBlocks(16,16,1);
    removeDupQuintupletsInGPU<<<dupBlocks,dupThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    hipDeviceSynchronize();
    free(threadIdx);
    free(nTriplets);
    hipFree(threadIdx_gpu);
#else
    printf("original 3D grid launching in createQuintuplets does not exist");
    exit(3);
#endif
#endif
    free(indicesOfEligibleModules);

#if defined(AddObjects)
#ifdef Explicit_T5
    addQuintupletsToEventExplicit();
#else
    addQuintupletsToEvent();
#endif
#endif

}

void SDL::Event::createTrackletsWithAGapWithModuleMap()
{
    //use the same trackletsInGPU as before if it exists
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#endif
    }

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsWithAGapInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }

}


void SDL::Event::addTrackletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(trackletsInGPU->nTracklets[i] == 0)
        {
            modulesInGPU->trackletRanges[idx * 2] = -1;
            modulesInGPU->trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            modulesInGPU->trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + trackletsInGPU->nTracklets[i] - 1;


            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
        }
    }
}
void SDL::Event::addTrackletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nTrackletsCPU;
hipHostMalloc(&nTrackletsCPU, nLowerModules * sizeof(unsigned int));
hipMemcpy(nTrackletsCPU,trackletsInGPU->nTracklets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_trackletRanges;
hipHostMalloc(&module_trackletRanges, nModules* 2*sizeof(int));
hipMemcpy(module_trackletRanges,modulesInGPU->trackletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTrackletsCPU[i] == 0)
        {
            module_trackletRanges[idx * 2] = -1;
            module_trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            module_trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + nTrackletsCPU[i] - 1;


            if(module_subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
        }
    }
hipHostFree(nTrackletsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_trackletRanges);
hipHostFree(module_layers);
}

void SDL::Event::addTrackCandidatesToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nTrackCandidatesCPU;
    hipHostMalloc(&nTrackCandidatesCPU, (nLowerModules )* sizeof(unsigned int));
    hipMemcpy(nTrackCandidatesCPU,trackCandidatesInGPU->nTrackCandidates,(nLowerModules)*sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_trackCandidateRanges;
    hipHostMalloc(&module_trackCandidateRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_trackCandidateRanges,modulesInGPU->trackCandidateRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);

    int* module_trackCandidateModuleIndices;
    hipHostMalloc(&module_trackCandidateModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMemcpy(module_trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, sizeof(int) * (nLowerModules + 1), hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];


        if(nTrackCandidatesCPU[i] == 0)
        {
            module_trackCandidateRanges[idx * 2] = -1;
            module_trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackCandidateRanges[idx * 2] = module_trackCandidateModuleIndices[i];
            module_trackCandidateRanges[idx * 2 + 1] = module_trackCandidateModuleIndices[i] + nTrackCandidatesCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
        }
    }
    hipHostFree(nTrackCandidatesCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_trackCandidateRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_trackCandidateModuleIndices);
}
void SDL::Event::addTrackCandidatesToEvent()
{

    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];


        if(trackCandidatesInGPU->nTrackCandidates[i] == 0 or SDL::modulesInGPU->trackCandidateModuleIndices[i] == -1)
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = -1;
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = SDL::modulesInGPU->trackCandidateModuleIndices[i];
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = SDL::modulesInGPU->trackCandidateModuleIndices[i] +  trackCandidatesInGPU->nTrackCandidates[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(quintupletsInGPU->nQuintuplets[i] == 0)
        {
            modulesInGPU->quintupletRanges[idx * 2] = -1;
            modulesInGPU->quintupletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->quintupletRanges[idx * 2] = SDL::modulesInGPU->quintupletModuleIndices[i];
            modulesInGPU->quintupletRanges[idx * 2 + 1] = SDL::modulesInGPU->quintupletModuleIndices[i] + quintupletsInGPU->nQuintuplets[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nQuintupletsCPU;
    hipHostMalloc(&nQuintupletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nQuintupletsCPU,quintupletsInGPU->nQuintuplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);

    int* module_quintupletRanges;
    hipHostMalloc(&module_quintupletRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_quintupletRanges,modulesInGPU->quintupletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    int* module_quintupletModuleIndices;
    hipHostMalloc(&module_quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpy(module_quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nQuintupletsCPU[i] == 0 or module_quintupletModuleIndices[i] == -1)
        {
            module_quintupletRanges[idx * 2] = -1;
            module_quintupletRanges[idx * 2 + 1] = -1;
        }
       else
        {
            module_quintupletRanges[idx * 2] = module_quintupletModuleIndices[i];
            module_quintupletRanges[idx * 2 + 1] = module_quintupletModuleIndices[i] + nQuintupletsCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[module_layers[idx] - 1] += nQuintupletsCPU[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[module_layers[idx] - 1] += nQuintupletsCPU[i];
            }
        }
    }
    hipHostFree(nQuintupletsCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_quintupletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_quintupletModuleIndices);

}

void SDL::Event::addTripletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            modulesInGPU->tripletRanges[idx * 2] = -1;
            modulesInGPU->tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            modulesInGPU->tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + tripletsInGPU->nTriplets[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
void SDL::Event::addTripletsToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nTripletsCPU;
    hipHostMalloc(&nTripletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_tripletRanges;
    hipHostMalloc(&module_tripletRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_tripletRanges,modulesInGPU->tripletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTripletsCPU[i] == 0)
        {
            module_tripletRanges[idx * 2] = -1;
            module_tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            module_tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + nTripletsCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
        }
    }
    hipHostFree(nTripletsCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_tripletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
}
#ifndef NESTED_PARA
__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;
    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

#ifdef NEWGRID_MD
    int lowerHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) / nUpperHits;
    int upperHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) % nUpperHits;
#else
    int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
#endif

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            #ifdef Warnings
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index =  %d\n",lowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif

        }

    }
}
#else
__global__ void createMiniDoubletsFromLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int lowerModuleIndex, unsigned int upperModuleIndex, unsigned int nLowerHits, unsigned int nUpperHits)
{
    unsigned int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);

        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            #ifdef Warnings
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index = %d\n",lowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif
        }

    }
}


__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;

    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(1,nLowerHits % nThreads.y == 0 ? nLowerHits/nThreads.y : nLowerHits/nThreads.y + 1, nUpperHits % nThreads.z == 0 ? nUpperHits/nThreads.z : nUpperHits/nThreads.z + 1);

    createMiniDoubletsFromLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, lowerModuleIndex, upperModuleIndex, nLowerHits, nUpperHits);


}
#endif

#ifndef NESTED_PARA
__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
#ifdef NEWGRID_Seg
    int innerLowerModuleArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;
    int outerLowerModuleArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
#else
    int xAxisIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int innerMDArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int outerMDArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;

    int innerLowerModuleArrayIdx = xAxisIdx/MAX_CONNECTED_MODULES;
    int outerLowerModuleArrayIdx = xAxisIdx % MAX_CONNECTED_MODULES; //need this index from the connected module array
#endif
    if(innerLowerModuleArrayIdx >= *modulesInGPU.nLowerModules) return;

    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIdx];

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];

    if(outerLowerModuleArrayIdx >= nConnectedModules) return;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIdx];

    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];
    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];

#ifdef NEWGRID_Seg
    if (nInnerMDs*nOuterMDs == 0) return;
    int innerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) / nOuterMDs;
    int outerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) % nOuterMDs;
#endif

    if(innerMDArrayIdx >= nInnerMDs) return;
    if(outerMDArrayIdx >= nOuterMDs) return;

    unsigned int innerMDIndex = modulesInGPU.mdRanges[innerLowerModuleIndex * 2] + innerMDArrayIdx;
    unsigned int outerMDIndex = modulesInGPU.mdRanges[outerLowerModuleIndex * 2] + outerMDArrayIdx;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            #ifdef Warnings
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, segmentIdx);
#endif

        }
    }
}
#else

__global__ void createSegmentsFromInnerLowerModule(struct SDL::modules&modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerLowerModuleIndex, unsigned int nInnerMDs)
{
    unsigned int outerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerMDArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int outerMDArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIndex];

    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];
    if(innerMDArrayIndex >= nInnerMDs) return;
    if(outerMDArrayIndex >= nOuterMDs) return;

    unsigned int innerMDIndex = innerLowerModuleIndex * N_MAX_MD_PER_MODULES + innerMDArrayIndex;
    unsigned int outerMDIndex = outerLowerModuleIndex * N_MAX_MD_PER_MODULES + outerMDArrayIndex;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);


    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            #ifdef Warnings
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, segmentIdx);
#endif

        }

    }

}

__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int innerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIndex];
    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];
    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];

    if(nConnectedModules == 0) return;

    if(nInnerMDs == 0) return;
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nConnectedModules % nThreads.x == 0 ? nConnectedModules/nThreads.x : nConnectedModules/nThreads.x + 1), (nInnerMDs % nThreads.y == 0 ? nInnerMDs/nThreads.y : nInnerMDs/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));

    createSegmentsFromInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerLowerModuleIndex,nInnerMDs);

}
#endif

#ifndef NESTED_PARA
#ifdef NEWGRID_Tracklet
__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int *index_gpu)
{
  //int innerInnerLowerModuleArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  if(nInnerSegments == 0) return;

  int outerInnerLowerModuleArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int innerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) % nInnerSegments;
  int outerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) / nInnerSegments;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  //number of possible outer segment inner MD lower modules
  unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
  if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

  unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

  //for completeness - outerOuterLowerModuleIndex
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  //with both segment indices obtained, run the tracklet algorithm
  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;

    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses


  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
      if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
      {
          #ifdef Warnings
          if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
              printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
          #endif
      }
      else
      {
          unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);

#endif

      }
    }
}
#endif
#else
__global__ void createTrackletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;
        //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;


    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //number of possible outer segment inner MD lower modules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;

    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);

#endif
        }
   }



}




__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
  int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
  if(nInnerSegments == 0) return;

  dim3 nThreads(1,16,16);
  dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

  createTrackletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}
#endif


#ifdef NEWGRID_Tracklet
__global__ void createTrackletsFromTriplets(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU,unsigned int *threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{

  int innerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex];

  if(nTriplets == 0) return;
  int innerTripletArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  int outerTripletArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x);

//////////////////////////////////////////////////////////
  if(innerTripletArrayIndex >= nTriplets) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner Triplet's outer lower module
  unsigned int innerTripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
  unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1]];//same as innerOuterInnerLowerModuleIndex
        if(outerTripletArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            unsigned int outerTripletIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
            unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
            unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];

            if(innerOuterSegmentIndex == outerInnerSegmentIndex)
            {
              unsigned int innerSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
              unsigned int outerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];
              unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
              unsigned int outerInnerLowerModuleIndex = segmentsInGPU.innerLowerModuleIndices[outerSegmentIndex];
              unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
              float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
              unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];

              float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
              bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

              if(success)
              {
                   unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
                   if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
                   {
                       #ifdef Warnings
                       if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                           printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
                       #endif
                   }
                   else
                   {
                       unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
                        #ifdef CUT_VALUE_DEBUG
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
                        #else
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
                        #endif
                   }
              }
            }
        }
}
#else
__global__ void createTrackletsFromTriplets(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU/*,unsigned int *index_gpu*/)
{
  int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex];

  if(nTriplets == 0) return;
    dim3 nThreads(16,16,1);
    dim3 nBlocks(nTriplets % nThreads.x == 0 ? nTriplets / nThreads.x : nTriplets / nThreads.x + 1, N_MAX_TRIPLETS_PER_MODULE % nThreads.y == 0 ? N_MAX_TRIPLETS_PER_MODULE / nThreads.y : N_MAX_TRIPLETS_PER_MODULE / nThreads.y + 1, 1);
    createTrackletsFromTripletsP2<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,trackletsInGPU,innerInnerLowerModuleArrayIndex,nTriplets);

}
__global__ void createTrackletsFromTripletsP2(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU/*,unsigned int *index_gpu*/,unsigned int innerInnerLowerModuleArrayIndex, unsigned int nTriplets)
{
  int innerTripletArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x);// % nTriplets;
  int outerTripletArrayIndex = (blockIdx.y * blockDim.y + threadIdx.y);// / nTriplets;
  if(innerTripletArrayIndex >= nTriplets) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner Triplet's outer lower module
  unsigned int innerTripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
  unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1]];//same as innerOuterInnerLowerModuleIndex
        if(outerTripletArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            unsigned int outerTripletIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
            unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
            unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];

            if(innerOuterSegmentIndex == outerInnerSegmentIndex)
            {
              unsigned int innerSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
              unsigned int outerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];
              unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
              unsigned int outerInnerLowerModuleIndex = segmentsInGPU.innerLowerModuleIndices[outerSegmentIndex];
              unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
              float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
              unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];

              float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
              bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

              if(success)
              {
                   unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
                   if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
                   {
                       #ifdef Warnings
                       if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                           printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
                       #endif
                   }
                   else
                   {
                       unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
                        #ifdef CUT_VALUE_DEBUG
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
                        #else
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
                        #endif
                   }
              }
            }
        }
}
#endif
#ifndef NESTED_PARA
__global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int* threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{
  int outerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex];
  unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
  unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
  unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];
  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(nOuterSegments == 0) return;
  if(nInnerSegments == 0) return;
  if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

  int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int outerSegmentArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  if(innerSegmentArrayIndex >= nInnerSegments) return;
  if(outerSegmentArrayIndex >= nOuterSegments) return;
  unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
  if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
  float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;

  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
  bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
      if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
	  if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
	    printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
      else
        {
	  unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
#else
      float eta = segmentsInGPU.eta[innerSegmentIndex];
      float phi = segmentsInGPU.phi[innerSegmentIndex];
      float pt = segmentsInGPU.ptIn[innerSegmentIndex];
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex,pt,eta,phi,0,0);
#endif
        }
    }
}

__device__ bool inline checkHits(unsigned int hit1, unsigned int hit2){

        if(hit1 == hit2){return true;}
        else {return false;}
        //float  x1 = hitsInGPU.xs[hit1];
        //float  y1 = hitsInGPU.ys[hit1];
        //float  z1 = hitsInGPU.zs[hit1];
        //float  x2 = hitsInGPU.xs[hit2];
        //float  y2 = hitsInGPU.ys[hit2];
        //float  z2 = hitsInGPU.zs[hit2];
        //float  dx = x1-x2;
        //float  dy = y1-y2;
        //float  dz = z1-z2;
        //float dR2 = dx*dx + dy*dy + dz*dz; 
        //if (dR2 < 0.0000000001){return true;}
        //else{ return false;}
}
__global__ void createPixelTrackletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex,unsigned int nInnerSegs,unsigned int* seg_pix_gpu, unsigned int* seg_pix_gpu_offset)
{
  //newgrid with map
  int segmentArrayIndex = seg_pix_gpu_offset[blockIdx.x * blockDim.x + threadIdx.x];// segment loop; this segent
  int pixelArrayIndex = seg_pix_gpu[blockIdx.x * blockDim.x + threadIdx.x];// pixel loop; this pixel
  if(pixelArrayIndex >= nInnerSegs) return;// don't exceed # of pLS
  if( segmentArrayIndex >= connectedPixelSize[pixelArrayIndex]) return; // don't exceed # connected segment modules for this pixel

  unsigned int outerInnerLowerModuleArrayIndex;// This will be the index of the module that connects to this pixel.
    unsigned int temp = connectedPixelIndex[pixelArrayIndex]+segmentArrayIndex; //gets module index for segment
    outerInnerLowerModuleArrayIndex = modulesInGPU.connectedPixels[temp]; //gets module index for segment
  if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int outerInnerLowerModuleIndex = /*modulesInGPU.lowerModuleIndices[*/outerInnerLowerModuleArrayIndex;//];

  unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
  unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(nOuterSegments == 0) return;
  if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

//  int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
  int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if(outerSegmentArrayIndex >= nOuterSegments) return;
  unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelArrayIndex;
  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
  if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
  float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;

  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
  bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
  if(success)
    {
//        printf("layer %d\n",modulesInGPU.layers[outerInnerLowerModuleIndex]);
        short layer2_adjustment;
        if(modulesInGPU.layers[outerInnerLowerModuleIndex] == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
        else if( modulesInGPU.layers[outerInnerLowerModuleIndex] == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
        else{return;} // ignore anything else
        float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex+layer2_adjustment]]]; 
        float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex+layer2_adjustment]]]; 
      float eta_pix = segmentsInGPU.eta[pixelArrayIndex];
      float phi_pix = segmentsInGPU.phi[pixelArrayIndex];
      //printf("diff %e %e %e %e\n",eta,eta1,phi,phi1);
      float pt = segmentsInGPU.ptIn[pixelArrayIndex];
    //int dup = -1;
//    for (unsigned int jx=0; jx<*pixelTrackletsInGPU.nPixelTracklets; jx++){
//        float eta1 = pixelTrackletsInGPU.eta[jx];
//        float phi1 = pixelTrackletsInGPU.phi[jx];
//        //float pt1 =  pixelTrackletsInGPU.pt[jx];
//        float dEta = abs(eta-eta1);
//        float dPhi = abs(phi-phi1);
//        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
//        //if( dEta> 0.03){continue;}
//        //if( dPhi > 0.03){continue;}
//        float dR2 = dEta*dEta + dPhi*dPhi; 
//        if(dR2 < 0.001){return;} // 0.0001(05) gives dup<1(2)% eff ~ 91(92)%
//        unsigned int hit1_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerSegmentIndex]];// inner seg (pixel) inner md inner hit
//        unsigned int hit1_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerSegmentIndex+1]];// inner seg (pixel) outer md inner hit
//        unsigned int hit1_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerSegmentIndex]+1];// inner seg inner md outer hit
//        unsigned int hit1_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerSegmentIndex+1]+1];// inner seg outer md outer hit
//        unsigned int hit1_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex]];// outer seg inner md inner hit
//        unsigned int hit1_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex+1]];// outer seg outer md inner hit
//        unsigned int hit1_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex]+1];// outer seg inner md outer hit
//        unsigned int hit1_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*outerSegmentIndex+1]+1];// outer seg outer md outer hit
//
//        unsigned int hit2_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]]]; // inner seg (pixel) inner md inner hit
//        unsigned int hit2_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]+1]];// inner seg outer md inner hit
//        unsigned int hit2_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]]+1]; // inner seg (pixel) inner md outer hit 
//        unsigned int hit2_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]+1]+1];// inner seg outer md outer hit
//        unsigned int hit2_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]]];// outer seg inner md inner hit
//        unsigned int hit2_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]+1]];// outer seg outer md innher hit 
//        unsigned int hit2_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]]+1];// outer seg inner md outer hit
//        unsigned int hit2_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]+1]+1];// outer seg outer md outer hit
//        //check pixel hits against each other
//        bool matched_11, matched_12, matched_13, matched_14;
//        matched_11 = checkHits(hit1_1,hit2_1);
//        matched_12 = checkHits(hit1_1,hit2_2);
//        matched_13 = checkHits(hit1_1,hit2_3);
//        matched_14 = checkHits(hit1_1,hit2_4);
//        bool matched_21, matched_22, matched_23, matched_24;
//        matched_21 = checkHits(hit1_2,hit2_1);
//        matched_22 = checkHits(hit1_2,hit2_2);
//        matched_23 = checkHits(hit1_2,hit2_3);
//        matched_24 = checkHits(hit1_2,hit2_4);
//        bool matched_31, matched_32, matched_33, matched_34;
//        matched_31 = checkHits(hit1_3,hit2_1);
//        matched_32 = checkHits(hit1_3,hit2_2);
//        matched_33 = checkHits(hit1_3,hit2_3);
//        matched_34 = checkHits(hit1_3,hit2_4);
//        bool matched_41, matched_42, matched_43, matched_44;
//        matched_41 = checkHits(hit1_4,hit2_1);
//        matched_42 = checkHits(hit1_4,hit2_2);
//        matched_43 = checkHits(hit1_4,hit2_3);
//        matched_44 = checkHits(hit1_4,hit2_4);
//        short matched_1 = matched_11 || matched_12 || matched_13 || matched_14;
//        short matched_2 = matched_21 || matched_22 || matched_23 || matched_24;
//        short matched_3 = matched_31 || matched_32 || matched_33 || matched_34;
//        short matched_4 = matched_41 || matched_42 || matched_43 || matched_44;
//        if (matched_1+matched_2+matched_3+matched_4 <= 2){
//        continue;}
//        bool matched_51, matched_52, matched_53, matched_54;
//        matched_51 = checkHits(hit1_5,hit2_5);
//        matched_52 = checkHits(hit1_5,hit2_6);
//        matched_53 = checkHits(hit1_5,hit2_7);
//        matched_54 = checkHits(hit1_5,hit2_8);
//        bool matched_61, matched_62, matched_63, matched_64;
//        matched_61 = checkHits(hit1_6,hit2_5);
//        matched_62 = checkHits(hit1_6,hit2_6);
//        matched_63 = checkHits(hit1_6,hit2_7);
//        matched_64 = checkHits(hit1_6,hit2_8);
//        bool matched_71, matched_72, matched_73, matched_74;
//        matched_71 = checkHits(hit1_7,hit2_5);
//        matched_72 = checkHits(hit1_7,hit2_6);
//        matched_73 = checkHits(hit1_7,hit2_7);
//        matched_74 = checkHits(hit1_7,hit2_8);
//        bool matched_81, matched_82, matched_83, matched_84;
//        matched_81 = checkHits(hit1_8,hit2_5);
//        matched_82 = checkHits(hit1_8,hit2_6);
//        matched_83 = checkHits(hit1_8,hit2_7);
//        matched_84 = checkHits(hit1_8,hit2_8);
//        short matched_5 = matched_51 || matched_52 || matched_53 || matched_54;
//        short matched_6 = matched_61 || matched_62 || matched_63 || matched_64;
//        short matched_7 = matched_71 || matched_72 || matched_73 || matched_74;
//        short matched_8 = matched_81 || matched_82 || matched_83 || matched_84;
//        if (matched_1+matched_2+matched_3+matched_4+matched_5+matched_6+matched_7+matched_8 < 7){
//        continue;} 
//        
//        return;
//    }
      unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
      if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
	  if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
	    printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
      else
        {
	  unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
#else
	    addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex,pt,eta,phi,eta_pix,phi_pix);
#endif
        }
    }
}
__device__ int duplicateCounter_TC =0;
__global__ void removeDupTrackCandidates(struct SDL::trackCandidates& trackCandidatesInGPU, struct SDL::modules& modulesInGPU)
{
    int dup_count=0;
    for(unsigned int lowmod1=blockIdx.x*blockDim.x+threadIdx.x; lowmod1<=*modulesInGPU.nLowerModules;lowmod1+=blockDim.x*gridDim.x){
      for(unsigned int ix1=blockIdx.y*blockDim.y+threadIdx.y; ix1<trackCandidatesInGPU.nTrackCandidates[lowmod1]; ix1+=blockDim.y*gridDim.y){
        bool isDup = false;
        unsigned int ix = modulesInGPU.trackCandidateModuleIndices[lowmod1] + ix1;
        if(trackCandidatesInGPU.isDup[ix]){continue;}
        float eta1 = trackCandidatesInGPU.eta[ix];
        float phi1 = trackCandidatesInGPU.phi[ix];
        for(unsigned int lowmod=0; lowmod<=*modulesInGPU.nLowerModules;lowmod++){
          for(unsigned int jx1=0; jx1<trackCandidatesInGPU.nTrackCandidates[lowmod]; jx1++){
            unsigned int jx = modulesInGPU.trackCandidateModuleIndices[lowmod] + jx1;
            if(ix>=jx){continue;}
            if(trackCandidatesInGPU.isDup[jx]){continue;}
            //float pt2  = trackCandidatesInGPU.pt[jx];
            float eta2 = trackCandidatesInGPU.eta[jx];
            float phi2 = trackCandidatesInGPU.phi[jx];
            float dEta = abs(eta1-eta2);
            float dPhi = abs(phi1-phi2);
            if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
            if (abs(eta1-eta2) > 0.08){continue;}
            if (abs(phi1-phi2) > 0.08){continue;}
            float dR2 = dEta*dEta + dPhi*dPhi; 
            if(dR2 < .005){
              isDup=true;break;
            }
          }
          if(isDup){break;}
        }
        if(isDup){rmTrackCandidateToMemory(trackCandidatesInGPU,ix);dup_count++;}
      }
  }
//  atomicAdd(&duplicateCounter_TC,dup_count);
//  printf("dup count: %d %d\n",dup_count,duplicateCounter_TC);
}
__device__ bool checkDupTrackCandidates(struct SDL::trackCandidates& trackCandidatesInGPU, float pt1, float eta1, float phi1)
{
 //   int dup_count=0;
    for (unsigned int jx=0; jx<*trackCandidatesInGPU.nTrackCandidates; jx++){
    //for (unsigned int jx=blockIdx.y*blockDim.y+threadIdx.y; jx<*pixelTrackletsInGPU.nPixelTracklets; jx+=blockDim.y*gridDim.y){
        //if(ix>=jx){continue;}
        //if(trackCandidatesInGPU.isDup[jx]){continue;}
        //float pt2  = trackCandidatesInGPU.pt[jx];
        float eta2 = trackCandidatesInGPU.eta[jx];
        float phi2 = trackCandidatesInGPU.phi[jx];
        float dEta = abs(eta1-eta2);
        float dPhi = abs(phi1-phi2);
        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
        //if (abs(eta1-eta2) > 0.03){continue;}
        //if (abs(phi1-phi2) > 0.03){continue;}
        float dR2 = dEta*dEta + dPhi*dPhi; 
        if(dR2 < 0.1){
          return true;//isDup=true;break;
        }
    }
    return false;
    //if(isDup){rmPixelTrackletToMemory(pixelTrackletsInGPU,ix);dup_count++;}
    //if(isDup){addPixelTrackletToMemory(pixelTrackletsInGPU,0,0,0,0,0,0,0,0,0,0,0,0,0,ix);}
//atomicAdd(&duplicateCounter_pT2,dup_count);
//printf("dup count: %d %d\n",dup_count,duplicateCounter_pT2);
}
__device__ int duplicateCounter_pT2 =0;
__global__ void removeDupPixelTrackletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU)
{
    int dup_count=0;
    //for (unsigned int ix=0; ix<*pixelTrackletsInGPU.nPixelTracklets; ix++){
    for (unsigned int ix=blockIdx.x*blockDim.x+threadIdx.x; ix<*pixelTrackletsInGPU.nPixelTracklets; ix+=blockDim.x*gridDim.x){
      bool isDup = false;
      if(pixelTrackletsInGPU.isDup[ix]){continue;}
//      float pt1 = pixelTrackletsInGPU.pt[ix];
      float eta1_pix = pixelTrackletsInGPU.eta_pix[ix]; 
      float phi1_pix = pixelTrackletsInGPU.phi_pix[ix]; 
      float eta1 = pixelTrackletsInGPU.eta[ix];
      float phi1 = pixelTrackletsInGPU.phi[ix];
      for (unsigned int jx=ix+1; jx<*pixelTrackletsInGPU.nPixelTracklets-1; jx++){
        if(pixelTrackletsInGPU.isDup[jx]){continue;}
        float eta2_pix = pixelTrackletsInGPU.eta_pix[jx]; 
        float phi2_pix = pixelTrackletsInGPU.phi_pix[jx]; 
        float dEta_pix = abs(eta1_pix-eta2_pix);
        float dPhi_pix = abs(phi1_pix-phi2_pix);
        if(dPhi_pix > M_PI){dPhi_pix = dPhi_pix - 2*M_PI;}
        if (dEta_pix > 0.03){continue;}
        if (dPhi_pix > 0.03){continue;}
        float dR2_pix = dEta_pix*dEta_pix + dPhi_pix*dPhi_pix; 
        if(dR2_pix < 0.001){
          isDup=true;break;
        }

        float eta2 = pixelTrackletsInGPU.eta[jx];
        float phi2 = pixelTrackletsInGPU.phi[jx];
        float dEta = abs(eta1-eta2);
        float dPhi = abs(phi1-phi2);
        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
        if (dEta > 0.03){continue;}
        if (dPhi > 0.03){continue;}
        float dR2 = dEta*dEta + dPhi*dPhi; 
        if(dR2 < 0.001){
          isDup=true;break;
        }
//        unsigned int hit1_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix]]];// inner seg (pixel) inner md inner hit
//        unsigned int hit1_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix]+1]];// inner seg (pixel) outer md inner hit
//        unsigned int hit1_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix]]+1];// inner seg inner md outer hit
//        unsigned int hit1_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix]+1]+1];// inner seg outer md outer hit
//        unsigned int hit1_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix+1]]];// outer seg inner md inner hit
//        unsigned int hit1_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix+1]+1]];// outer seg outer md inner hit
//        unsigned int hit1_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix+1]]+1];// outer seg inner md outer hit
//        unsigned int hit1_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*ix+1]+1]+1];// outer seg outer md outer hit
//
//        unsigned int hit2_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]]]; // inner seg (pixel) inner md inner hit
//        unsigned int hit2_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]+1]];// inner seg outer md inner hit
//        unsigned int hit2_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]]+1]; // inner seg (pixel) inner md outer hit 
//        unsigned int hit2_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx]+1]+1];// inner seg outer md outer hit
//        unsigned int hit2_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]]];// outer seg inner md inner hit
//        unsigned int hit2_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]+1]];// outer seg outer md innher hit 
//        unsigned int hit2_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]]+1];// outer seg inner md outer hit
//        unsigned int hit2_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTrackletsInGPU.segmentIndices[2*jx+1]+1]+1];// outer seg outer md outer hit
//        //check pixel hits against each other
//        bool matched_11, matched_12, matched_13, matched_14;
//        matched_11 = checkHits(hit1_1,hit2_1);
//        matched_12 = checkHits(hit1_1,hit2_2);
//        matched_13 = checkHits(hit1_1,hit2_3);
//        matched_14 = checkHits(hit1_1,hit2_4);
//        bool matched_21, matched_22, matched_23, matched_24;
//        matched_21 = checkHits(hit1_2,hit2_1);
//        matched_22 = checkHits(hit1_2,hit2_2);
//        matched_23 = checkHits(hit1_2,hit2_3);
//        matched_24 = checkHits(hit1_2,hit2_4);
//        bool matched_31, matched_32, matched_33, matched_34;
//        matched_31 = checkHits(hit1_3,hit2_1);
//        matched_32 = checkHits(hit1_3,hit2_2);
//        matched_33 = checkHits(hit1_3,hit2_3);
//        matched_34 = checkHits(hit1_3,hit2_4);
//        bool matched_41, matched_42, matched_43, matched_44;
//        matched_41 = checkHits(hit1_4,hit2_1);
//        matched_42 = checkHits(hit1_4,hit2_2);
//        matched_43 = checkHits(hit1_4,hit2_3);
//        matched_44 = checkHits(hit1_4,hit2_4);
//        short matched_1 = matched_11 || matched_12 || matched_13 || matched_14;
//        short matched_2 = matched_21 || matched_22 || matched_23 || matched_24;
//        short matched_3 = matched_31 || matched_32 || matched_33 || matched_34;
//        short matched_4 = matched_41 || matched_42 || matched_43 || matched_44;
//        //if (matched_1+matched_2+matched_3+matched_4 <= 2){
//        //continue;}
//        bool matched_51, matched_52, matched_53, matched_54;
//        matched_51 = checkHits(hit1_5,hit2_5);
//        matched_52 = checkHits(hit1_5,hit2_6);
//        matched_53 = checkHits(hit1_5,hit2_7);
//        matched_54 = checkHits(hit1_5,hit2_8);
//        bool matched_61, matched_62, matched_63, matched_64;
//        matched_61 = checkHits(hit1_6,hit2_5);
//        matched_62 = checkHits(hit1_6,hit2_6);
//        matched_63 = checkHits(hit1_6,hit2_7);
//        matched_64 = checkHits(hit1_6,hit2_8);
//        bool matched_71, matched_72, matched_73, matched_74;
//        matched_71 = checkHits(hit1_7,hit2_5);
//        matched_72 = checkHits(hit1_7,hit2_6);
//        matched_73 = checkHits(hit1_7,hit2_7);
//        matched_74 = checkHits(hit1_7,hit2_8);
//        bool matched_81, matched_82, matched_83, matched_84;
//        matched_81 = checkHits(hit1_8,hit2_5);
//        matched_82 = checkHits(hit1_8,hit2_6);
//        matched_83 = checkHits(hit1_8,hit2_7);
//        matched_84 = checkHits(hit1_8,hit2_8);
//        short matched_5 = matched_51 || matched_52 || matched_53 || matched_54;
//        short matched_6 = matched_61 || matched_62 || matched_63 || matched_64;
//        short matched_7 = matched_71 || matched_72 || matched_73 || matched_74;
//        short matched_8 = matched_81 || matched_82 || matched_83 || matched_84;
//        if (matched_1+matched_2+matched_3+matched_4+matched_5+matched_6+matched_7+matched_8 >= 6){
//          isDup=true;break;
//        }
    }
    if(isDup){rmPixelTrackletToMemory(pixelTrackletsInGPU,ix);dup_count++;}
    //if(isDup){addPixelTrackletToMemory(pixelTrackletsInGPU,0,0,0,0,0,0,0,0,0,0,0,0,0,ix);}
  } 
//atomicAdd(&duplicateCounter_pT2,dup_count);
//printf("dup count: %d %d\n",dup_count,duplicateCounter_pT2);
}

#else
__global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; // loop for modules for segments lower hit
    if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return; // don't exceed number of modules

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex]; // correspond to module number index
    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; // pixel module index
    unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
    unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex]; // number of pLS
    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex]; // number of segments from module corresponding to each module.
    if(nOuterSegments == 0) return;
    if(nInnerSegments == 0) return;
    if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, nOuterSegments % nThreads.y == 0 ? nOuterSegments / nThreads.y : nOuterSegments / nThreads.y + 1, 1);

    createPixelTrackletsFromOuterInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelTrackletsInGPU, outerInnerLowerModuleIndex, nInnerSegments, nOuterSegments, pixelModuleIndex, pixelLowerModuleArrayIndex);

}
__global__ void createPixelTrackletsFromOuterInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int outerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nOuterSegments, unsigned int pixelModuleIndex, unsigned int pixelLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;// looping over pixels
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;//looping over segments
    if(innerSegmentArrayIndex >= nInnerSegments) return; // not over # of pLS
    if(outerSegmentArrayIndex >= nOuterSegments) return; // not over # of segments for this module
    unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex; // get this pixel index Just innerSegmentArrayIndex'th value (1-pLS)
    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex; // get this segment Index for this this module
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex]; // get corresponding outer module index for this segment
    if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

    if(success)
    {
        unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
        if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
                printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
        else
        {
            unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
                addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
      float eta = segmentsInGPU.eta[innerSegmentIndex];
      float phi = segmentsInGPU.eta[innerSegmentIndex];
      float pt = segmentsInGPU.eta[innerSegmentIndex];
            addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex,pt,eta,phi,0,0);
#endif
        }
    }
}
#endif

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Proposal 1 : Inner kernel takes care of both loops
    int xAxisIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex =  blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    int middleLowerModuleArrayIndex = xAxisIndex / MAX_CONNECTED_MODULES;
    int outerInnerLowerModuleArrayIndex = xAxisIndex % MAX_CONNECTED_MODULES;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //first check for middle modules
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    //second check for outerInnerLowerMoules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut, pt_beta;
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                 printf("T4x excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {

            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
#endif

        }
   }
}

__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 1
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) % nThreads.x  == 0 ? (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x : (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

#ifndef NESTED_PARA
#ifdef NEWGRID_Trips
__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int *index_gpu)
{
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
  if(nConnectedModules == 0) return;

  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int outerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  //middle lower module - outer lower module of inner segment
  unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut, pt_beta;
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

  if(success)
    {
      unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
      if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
      {
          #ifdef Warnings
          if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
              printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
          #endif
      }
      unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut,pt_beta, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
      addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, betaIn, betaOut, pt_beta, tripletIndex);
#endif
    }
}
#endif
#else
__global__ void createTripletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nConnectedModules, unsigned int innerInnerLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - outer lower module of inner segment
    unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;
    unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    if(success)
    {
        unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
        if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
                printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

            addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, betaIn, betaOut, pt_beta, tripletIndex);
#endif

        }
    }
}

__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex] ;
    if(nInnerSegments == 0) return;

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
    if(nConnectedModules == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE / nThreads.y : N_MAX_SEGMENTS_PER_MODULE / nThreads.y + 1);

    createTripletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, nConnectedModules, innerInnerLowerModuleArrayIndex);
}
#endif

__global__ void addT5asTrackCandidateInGPU(struct SDL::modules& modulesInGPU,struct SDL::quintuplets& quintupletsInGPU,struct SDL::trackCandidates& trackCandidatesInGPU)
{

  int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules or modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1) return; 
  unsigned int nQuints = quintupletsInGPU.nQuintuplets[innerInnerInnerLowerModuleArrayIndex];
  if (nQuints > N_MAX_QUINTUPLETS_PER_MODULE) {nQuints = N_MAX_QUINTUPLETS_PER_MODULE;}
  int innerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if(innerObjectArrayIndex >= nQuints) return;
  int quintupletIndex = modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] + innerObjectArrayIndex;
  if(quintupletsInGPU.isDup[quintupletIndex]){return;} 
  float pt = quintupletsInGPU.pt[quintupletIndex];
  float eta = quintupletsInGPU.eta[quintupletIndex];
  float phi = quintupletsInGPU.phi[quintupletIndex];
//  if(checkDupTrackCandidates(trackCandidatesInGPU,pt,eta,phi)){return;} 

 
  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
  atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT5[innerInnerInnerLowerModuleArrayIndex],1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
//  printf("T5 %e %e %e\n",pt,eta,phi);
  addTrackCandidateToMemory(trackCandidatesInGPU, 4/*track candidate type T5=4*/, quintupletIndex, quintupletIndex, trackCandidateIdx,pt,eta,phi);
}

__global__ void addpT2asTrackCandidateInGPU(struct SDL::modules& modulesInGPU,struct SDL::pixelTracklets& pixelTrackletsInGPU,struct SDL::trackCandidates& trackCandidatesInGPU)
{
  int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;
  unsigned int nPixelTracklets = *pixelTrackletsInGPU.nPixelTracklets; 
  if(pixelTrackletArrayIndex >= nPixelTracklets) return;
  int pixelTrackletIndex = pixelTrackletArrayIndex;
  if(pixelTrackletsInGPU.isDup[pixelTrackletIndex]){return;} 
  float pt  = pixelTrackletsInGPU.pt[pixelTrackletIndex];
  float eta = pixelTrackletsInGPU.eta[pixelTrackletIndex];
  float phi = pixelTrackletsInGPU.phi[pixelTrackletIndex];
//  if(checkDupTrackCandidates(trackCandidatesInGPU,pt,eta,phi)){return;} 
  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
  atomicAdd(trackCandidatesInGPU.nTrackCandidatespT2,1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
//  printf("pT2 %e %e %e\n",pt,eta,phi);
  addTrackCandidateToMemory(trackCandidatesInGPU, 3/*track candidate type pT2=3*/, pixelTrackletIndex, pixelTrackletIndex, trackCandidateIdx,pt,eta,phi);
}

__global__ void addpT3asTrackCandidateInGPU(struct SDL::modules& modulesInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
  int pixelTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;
  unsigned int nPixelTriplets = *pixelTripletsInGPU.nPixelTriplets; 
  if(pixelTripletArrayIndex >= nPixelTriplets) return;
  int pixelTripletIndex = pixelTripletArrayIndex;
  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
  atomicAdd(trackCandidatesInGPU.nTrackCandidatespT3,1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
  addTrackCandidateToMemory(trackCandidatesInGPU, 5/*track candidate type pT3=5*/, pixelTripletIndex, pixelTripletIndex, trackCandidateIdx,0,0,0);

}


#ifndef NESTED_PARA
__global__ void createPixelTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int* threadIdx_gpu, unsigned int* threadIdx_gpu_offset)
{
  unsigned int outerInnerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(outerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  //FIXME:Cheapo module map - We care about pT4s and pTCs Only if the outerInnerInnerLowerModule is "connected" to the pixel module

  int outerInnerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerInnerLowerModuleArrayIndex];
  if(modulesInGPU.moduleType[outerInnerInnerLowerModuleIndex] == SDL::TwoS) return;

  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;

  unsigned int nPixelTracklets = *(pixelTrackletsInGPU.nPixelTracklets);
  //capping
  if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
    nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;

  unsigned int nOuterLayerTracklets = trackletsInGPU.nTracklets[outerInnerInnerLowerModuleArrayIndex];
  if(nOuterLayerTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
      nOuterLayerTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
  unsigned int nOuterLayerTriplets = tripletsInGPU.nTriplets[outerInnerInnerLowerModuleArrayIndex];
  if(nOuterLayerTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
      nOuterLayerTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

  unsigned int nThreadsForNestedKernel = max(nOuterLayerTracklets,nOuterLayerTriplets);
  if(nThreadsForNestedKernel == 0) return;

  int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  int outerObjectArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y+ threadIdx.y];
  if(pixelTrackletArrayIndex >= nPixelTracklets) return;

  int pixelTrackletIndex = pixelTrackletArrayIndex;
  int outerObjectIndex = 0;
  short trackCandidateType;
  bool success;

  //pT4-T4
  if(outerObjectArrayIndex < nOuterLayerTracklets)
    {
      outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
        if(modulesInGPU.moduleType[trackletsInGPU.lowerModuleIndices[4 * outerObjectIndex + 1]] == SDL::PS)
        {
	        success = runTrackCandidateDefaultAlgoTwoTracklets(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
	    if(success)
        {
	        unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
	        atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
	        if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
            {
                #ifdef Warnings
    		  if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {

		            printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
                    #endif
            }
	        else
            {
		    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
            {
                #ifdef Warnings
		        printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                #endif

            }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, 5/*trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx,0,0,0);
                    }

                }
            }
        }
    }

  //pT4-T3
  if(outerObjectArrayIndex < nOuterLayerTriplets)
    {
      outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;

      //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
      if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1]] == SDL::PS)
        {
	  success = runTrackCandidateDefaultAlgoTrackletToTriplet(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
		      printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, 6/*trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx,0,0,0);
                    }

                }
            }
        }

    }
}

#else
__global__ void createPixelTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
    unsigned int outerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(outerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    int outerInnerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerInnerLowerModuleArrayIndex];
    if(modulesInGPU.moduleType[outerInnerInnerLowerModuleIndex] == SDL::TwoS) return;

    unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;

    unsigned int nPixelTracklets = *(pixelTrackletsInGPU.nPixelTracklets);
    //capping
    if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;

    unsigned int nOuterLayerTracklets = trackletsInGPU.nTracklets[outerInnerInnerLowerModuleArrayIndex];
    if(nOuterLayerTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
        nOuterLayerTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
    unsigned int nOuterLayerTriplets = tripletsInGPU.nTriplets[outerInnerInnerLowerModuleArrayIndex];
    if(nOuterLayerTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
        nOuterLayerTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

    unsigned int nThreadsForNestedKernel = max(nOuterLayerTracklets,nOuterLayerTriplets);
    if(nThreadsForNestedKernel == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks( nPixelTracklets % nThreads.x == 0 ? nPixelTracklets/nThreads.x : nPixelTracklets/nThreads.x + 1, nThreadsForNestedKernel % nThreads.y == 0 ? nThreadsForNestedKernel/nThreads.y : nThreadsForNestedKernel/nThreads.y + 1, 1);

    createPixelTrackCandidatesFromOuterInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, pixelTrackletsInGPU, tripletsInGPU, trackCandidatesInGPU, pixelLowerModuleArrayIndex, outerInnerInnerLowerModuleArrayIndex, nPixelTracklets, nOuterLayerTracklets, nOuterLayerTriplets);
}


__global__ void createPixelTrackCandidatesFromOuterInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int pixelLowerModuleArrayIndex, unsigned int outerInnerInnerLowerModuleArrayIndex, unsigned int nPixelTracklets, unsigned int nOuterLayerTracklets, unsigned int nOuterLayerTriplets)
{
    int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(pixelTrackletArrayIndex >= nPixelTracklets) return;

    int pixelTrackletIndex = pixelTrackletArrayIndex;
    int outerObjectIndex = 0;
    short trackCandidateType;
    bool success;

    //pT4-T4
    if(outerObjectArrayIndex < nOuterLayerTracklets)
    {
        outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
       if(modulesInGPU.moduleType[trackletsInGPU.lowerModuleIndices[4 * outerObjectIndex + 1]] == SDL::PS)
        {
            success = runTrackCandidateDefaultAlgoTwoTracklets(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, 5/*pT2-T4 trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }
    }

    //pT4-T3
    if(outerObjectArrayIndex < nOuterLayerTriplets)
    {
        outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
        if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1]] == SDL::PS)
        {
            success = runTrackCandidateDefaultAlgoTrackletToTriplet(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, 6/* pT2-T3 trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }

    }
}
#endif

#ifndef NESTED_PARA
__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int* threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{
  //inner tracklet/triplet inner segment inner MD lower module
  int innerInnerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  //hack to include pixel detector
  if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
  if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
      nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }

  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
  if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
      nTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

  unsigned int temp = max(nTracklets,nTriplets);
  unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

  if(temp == 0) return;

  int innerObjectArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  int outerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  int innerObjectIndex = 0;
  int outerObjectIndex = 0;
  short trackCandidateType;
  bool success;

  //step 1 tracklet-tracklet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

      if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

	  success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx,0,0,0);
                    }

                }
            }

        }
    }

  //step 2 tracklet-triplet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
      if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
	  success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {

		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                        #endif
                    }
		  else
                    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx,0,0,0);
                    }
                }
            }

        }
    }
  //step 3 triplet-tracklet
  if(innerObjectArrayIndex < nTriplets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//same as innerOuterInnerLowerModuleIndex

      if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {
	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
	  success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                   #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
		    printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                   #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d, outer T4 module index = %d\n",innerInnerInnerLowerModuleArrayIndex, outerInnerInnerLowerModuleIndex);
                        #endif
                    }
		  else
                    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx,0,0,0);

                    }
                }
            }

        }
    }
}

#else
__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
    //inner tracklet/triplet inner segment inner MD lower module
    int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //hack to include pixel detector
    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
    if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
        nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }

    unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
    if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
        nTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

    unsigned int temp = max(nTracklets,nTriplets);
    unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

    if(temp == 0) return;

    //triplets and tracklets are stored directly using lower module array index
    dim3 nThreads(16,16,1);
    dim3 nBlocks(temp % nThreads.x == 0 ? temp / nThreads.x : temp / nThreads.x + 1, MAX_OBJECTS % nThreads.y == 0 ? MAX_OBJECTS / nThreads.y : MAX_OBJECTS / nThreads.y + 1, 1);

    createTrackCandidatesFromInnerInnerInnerLowerModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, tripletsInGPU, trackCandidatesInGPU,innerInnerInnerLowerModuleArrayIndex,nTracklets,nTriplets);
}

__global__ void createTrackCandidatesFromInnerInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int innerInnerInnerLowerModuleArrayIndex, unsigned int nInnerTracklets, unsigned int nInnerTriplets)
{
    int innerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int innerObjectIndex = 0;
    int outerObjectIndex = 0;
    short trackCandidateType;
    bool success;
    //step 1 tracklet-tracklet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

            success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }

        }
    }
    //step 2 tracklet-triplet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
        if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {

                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
                        printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                        #endif
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }
                }
            }

        }
    }

    //step 3 triplet-tracklet
    if(innerObjectArrayIndex < nInnerTriplets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//same as innerOuterInnerLowerModuleIndex

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	        if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                   #ifdef Warnings
                   if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                       printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                   #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
                        printf("Track candidates: no memory for module at module index = %d, outer T4 module index = %d\n",innerInnerInnerLowerModuleArrayIndex, outerInnerInnerLowerModuleIndex);
                        #endif
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

                    }
                }
            }

        }
    }
}
#endif


//#ifndef NESTED_PARA
//#else

__global__ void createPixelTripletsFromOuterInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, unsigned int outerTripletInnerLowerModuleArrayIndex, unsigned int nPixelSegments, unsigned int nOuterTriplets, unsigned int pixelModuleIndex)
{
   int pixelSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
   int outerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

   if(pixelSegmentArrayIndex >= nPixelSegments) return;
   if(outerTripletArrayIndex >= nOuterTriplets) return;

   unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelSegmentArrayIndex;
   unsigned int outerTripletIndex = outerTripletInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;

   if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1]] == SDL::TwoS) return; //REMOVES PS-2S

   float pixelRadius, pixelRadiusError, tripletRadius;
   bool success = runPixelTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius);

   if(success)
   {
       unsigned int pixelTripletIndex = atomicAdd(pixelTripletsInGPU.nPixelTriplets, 1);
       if(pixelTripletIndex >= N_MAX_PIXEL_TRIPLETS)
       {
            #ifdef Warnings
            if(pixelTripletIndex == N_MAX_PIXEL_TRIPLETS)
            {
               printf("Pixel Triplet excess alert!\n"); 
            }
            #endif
       }
       else
       {
#ifdef CUT_VALUE_DEBUG
           addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, pixelTripletIndex);
#else
           addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius,tripletRadius, pixelTripletIndex);
#endif
       }
   }
}

__global__ void createPixelTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU)
{
    int outerTripletInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

    //lower modules 2 and 3 are taken from the triplet!
    if(outerTripletInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int nOuterTriplets = tripletsInGPU.nTriplets[outerTripletInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[outerTripletInnerLowerModuleArrayIndex];

    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1;
    unsigned int nPixelSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];
    
    //El-cheapo map applied on the inner segment
    unsigned int outerTripletInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerTripletInnerLowerModuleArrayIndex];
    if(modulesInGPU.moduleType[outerTripletInnerLowerModuleIndex]== SDL::TwoS) return; //REMOVES 2S-2S

    if(nOuterTriplets == 0) return;
    dim3 nThreads(16,16,1);
    dim3 nBlocks(nPixelSegments % nThreads.x == 0 ? nPixelSegments / nThreads.x : nPixelSegments / nThreads.x + 1, nOuterTriplets % nThreads.y == 0 ? nOuterTriplets / nThreads.y : nOuterTriplets / nThreads.y + 1, 1);

    createPixelTripletsFromOuterInnerLowerModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelTripletsInGPU, outerTripletInnerLowerModuleArrayIndex, nPixelSegments, nOuterTriplets, pixelModuleIndex);
}

//#endif

__device__ int duplicateCounter;
__global__ void removeDupQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int* threadIdx_gpu, unsigned int* threadIdx_gpu_offset)
{
      int dup_count=0;
      for(unsigned int lowmod1=blockIdx.x*blockDim.x+threadIdx.x; lowmod1<*modulesInGPU.nLowerModules;lowmod1+=blockDim.x*gridDim.x){
      for(unsigned int ix1=blockIdx.y*blockDim.y+threadIdx.y; ix1<quintupletsInGPU.nQuintuplets[lowmod1]; ix1+=blockDim.y*gridDim.y){
        unsigned int ix = modulesInGPU.quintupletModuleIndices[lowmod1] + ix1;
        if(quintupletsInGPU.isDup[ix]==1){continue;}
        float pt1  = quintupletsInGPU.pt[ix];
        float eta1 = quintupletsInGPU.eta[ix];
        float phi1 = quintupletsInGPU.phi[ix];
        bool isDup = false;
        for(unsigned int lowmod=0; lowmod<*modulesInGPU.nLowerModules;lowmod++){
      for(unsigned int jx1=0; jx1<quintupletsInGPU.nQuintuplets[lowmod]; jx1++){
        unsigned int jx = modulesInGPU.quintupletModuleIndices[lowmod] + jx1;
        if(ix>=jx){continue;}
        //if(ix==jx){continue;}
        //for(unsigned int jx=0; jx<quintupletsInGPU.nQuintuplets[lowmod]+modulesInGPU.quintupletModuleIndices[lowmod]; jx++){
        if(quintupletsInGPU.isDup[jx]==1){continue;}
        float pt2  = quintupletsInGPU.pt[jx];
        float eta2 = quintupletsInGPU.eta[jx];
        float phi2 = quintupletsInGPU.phi[jx];
        float dEta = abs(eta1-eta2);
        float dPhi = abs(phi1-phi2);
        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
        if (dEta > 0.5){continue;}
        if (dPhi > 0.5){continue;}
        float dR2 = dEta*dEta + dPhi*dPhi;
        if(dR2 < 0.001){
          isDup=true;break;
        }
        unsigned int hit1_0 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]]]; // inner triplet inner segment inner md inner hit
        unsigned int hit1_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]]+1]; // inner triplet inner segment inner md outer hit
        unsigned int hit1_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]+1]]; // inner triplet inner segment outer md inner hit
        unsigned int hit1_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]+1]+1]; // inner triplet inner segment outer md outer hit
        unsigned int hit1_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]+1]+1]]; // inner triplet outer segment outer md inner hit
        unsigned int hit1_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]+1]+1]+1]; // inner triplet outer segment outer md outer hit
        unsigned int hit1_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]]]; // outer triplet outersegment inner md inner hit 
        unsigned int hit1_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]]+1]; // outer triplet outersegment inner md outer hit
        unsigned int hit1_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]+1]]; // outer triplet outersegment outer md inner hit
        unsigned int hit1_9 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]+1]+1]; // outer triplet outersegment outer md outer hit

        unsigned int hit2_0 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]]; // inner triplet inner segment inner md inner hit
        unsigned int hit2_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]+1]; // inner triplet inner segment inner md outer hit
        unsigned int hit2_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]]; // inner triplet inner segment outer md inner hit
        unsigned int hit2_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]+1]; // inner triplet inner segment outer md outer hit
        unsigned int hit2_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]]; // inner triplet outer segment outer md inner hit
        unsigned int hit2_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]+1]; // inner triplet outer segment outer md outer hit
        unsigned int hit2_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]]; // outer triplet outersegment inner md inner hit 
        unsigned int hit2_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]+1]; // outer triplet outersegment inner md outer hit
        unsigned int hit2_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]]; // outer triplet outersegment outer md inner hit
        unsigned int hit2_9 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]+1]; // outer triplet outersegment outer md outer hit

        short matched_0 = checkHits(hit1_0,hit2_0) ||checkHits(hit1_0,hit2_2); 
        short matched_1 = checkHits(hit1_1,hit2_1) ||checkHits(hit1_1,hit2_3);
        short matched_2 = checkHits(hit1_2,hit2_0) ||checkHits(hit1_2,hit2_2) ||checkHits(hit1_2,hit2_4);
        short matched_3 = checkHits(hit1_3,hit2_1) ||checkHits(hit1_3,hit2_3) ||checkHits(hit1_3,hit2_5);
        short matched_4 = checkHits(hit1_4,hit2_2) ||checkHits(hit1_4,hit2_4) ||checkHits(hit1_4,hit2_6);
        short matched_5 = checkHits(hit1_5,hit2_3) ||checkHits(hit1_5,hit2_5) ||checkHits(hit1_5,hit2_7);
        short matched_6 = checkHits(hit1_6,hit2_4) ||checkHits(hit1_6,hit2_6) ||checkHits(hit1_6,hit2_8);
        short matched_7 = checkHits(hit1_7,hit2_5) ||checkHits(hit1_7,hit2_7) ||checkHits(hit1_7,hit2_9);
        short matched_8 = checkHits(hit1_8,hit2_6) ||checkHits(hit1_8,hit2_8);
        short matched_9 = checkHits(hit1_9,hit2_7) ||checkHits(hit1_9,hit2_9);

        if(matched_0+matched_1+matched_2+matched_3+matched_4+matched_5+matched_6+matched_7+matched_8+matched_9>7){
                isDup=true;//dup_count++;
//                printf("hits1 %u %u %u %u %u %u %u %u %u %u\n",hit1_0,hit1_1,hit1_2,hit1_3,hit1_4,hit1_5,hit1_6,hit1_7,hit1_8,hit1_9);
//                printf("hits2 %u %u %u %u %u %u %u %u %u %u\n",hit2_0,hit2_1,hit2_2,hit2_3,hit2_4,hit2_5,hit2_6,hit2_7,hit2_8,hit2_9);
//                printf("id %u %d\n",ix,dup_count);
                break;
          }
      }
      if(isDup){break;}
      }
                //if(isDup){dup_count++;addQuintupletToMemory(quintupletsInGPU, 0, 0, 0, 0, 0, 0,0, 0, 0, ix,1,pt1,eta1,phi1);}
                if(isDup){dup_count++;rmQuintupletToMemory(quintupletsInGPU,ix);}
    }}
//atomicAdd(&duplicateCounter,dup_count);
//printf("dup count: %d %d\n",dup_count,duplicateCounter);

}
#ifndef NESTED_PARA
__global__ void createQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int* threadIdx_gpu, unsigned int* threadIdx_gpu_offset)
{
    int lowerModuleArray1 = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];

    //this if statement never gets executed!
    if(lowerModuleArray1  >= *modulesInGPU.nLowerModules) return;

    unsigned int nInnerTriplets = tripletsInGPU.nTriplets[lowerModuleArray1];

    //unsigned int innerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int innerTripletArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
    unsigned int outerTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(innerTripletArrayIndex >= nInnerTriplets) return;

    unsigned int innerTripletIndex = lowerModuleArray1 * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
    unsigned int lowerModule1 = modulesInGPU.lowerModuleIndices[lowerModuleArray1];
    //these are actual module indices!! not lower module indices!
    unsigned int lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
    unsigned int lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
    unsigned int lowerModuleArray3 = modulesInGPU.reverseLookupLowerModuleIndices[lowerModule3];
    unsigned int nOuterTriplets = min(tripletsInGPU.nTriplets[lowerModuleArray3], N_MAX_TRIPLETS_PER_MODULE);

    if(outerTripletArrayIndex >= nOuterTriplets) return;
    unsigned int outerTripletIndex = lowerModuleArray3 * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
    //these are actual module indices!!
    unsigned int lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
    unsigned int lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

    float innerRadius, innerRadiusMin, innerRadiusMin2S, innerRadiusMax, innerRadiusMax2S, outerRadius, outerRadiusMin, outerRadiusMin2S, outerRadiusMax, outerRadiusMax2S, bridgeRadius, bridgeRadiusMin, bridgeRadiusMin2S, bridgeRadiusMax, bridgeRadiusMax2S; //required for making distributions
    bool success = runQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S,
            outerRadiusMax2S);

   if(success)
   {
//      for(unsigned int lowmod=0; lowmod<*modulesInGPU.nLowerModules;lowmod++){
//      for(unsigned int jx1=0; jx1<quintupletsInGPU.nQuintuplets[lowmod]; jx1++){
//        unsigned int jx = modulesInGPU.quintupletModuleIndices[lowmod] + jx1;
//      //for(unsigned int jx=0; jx<quintupletsInGPU.nQuintuplets[lowmod]+modulesInGPU.quintupletModuleIndices[lowmod]; jx++){
//        //printf("jx: %u \n",jx);
//        unsigned int hit1_0 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]]]; // inner triplet inner segment inner md inner hit
//        unsigned int hit1_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]]+1]; // inner triplet inner segment inner md outer hit
//        unsigned int hit1_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]+1]]; // inner triplet inner segment outer md inner hit
//        unsigned int hit1_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]+1]+1]; // inner triplet inner segment outer md outer hit
//        unsigned int hit1_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+1]+1]]; // inner triplet outer segment outer md inner hit
//        unsigned int hit1_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+1]+1]+1]; // inner triplet outer segment outer md outer hit
//        unsigned int hit1_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]]]; // outer triplet outersegment inner md inner hit 
//        unsigned int hit1_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]]+1]; // outer triplet outersegment inner md outer hit
//        unsigned int hit1_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]+1]]; // outer triplet outersegment outer md inner hit
//        unsigned int hit1_9 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]+1]+1]; // outer triplet outersegment outer md outer hit
//
//        unsigned int hit2_0 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]]; // inner triplet inner segment inner md inner hit
//        unsigned int hit2_1 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]+1]; // inner triplet inner segment inner md outer hit
//        unsigned int hit2_2 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]]; // inner triplet inner segment outer md inner hit
//        unsigned int hit2_3 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]+1]; // inner triplet inner segment outer md outer hit
//        unsigned int hit2_4 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]]; // inner triplet outer segment outer md inner hit
//        unsigned int hit2_5 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]+1]; // inner triplet outer segment outer md outer hit
//        unsigned int hit2_6 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]]; // outer triplet outersegment inner md inner hit 
//        unsigned int hit2_7 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]+1]; // outer triplet outersegment inner md outer hit
//        unsigned int hit2_8 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]]; // outer triplet outersegment outer md inner hit
//        unsigned int hit2_9 = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]+1]; // outer triplet outersegment outer md outer hit
//
//        //short matched_0 = checkHits(hit1_0,hit2_0) ||checkHits(hit1_0,hit2_1) ||checkHits(hit1_0,hit2_2) ||checkHits(hit1_0,hit2_3) ||checkHits(hit1_0,hit2_4) ||checkHits(hit1_0,hit2_5) ||checkHits(hit1_0,hit2_6) ||checkHits(hit1_0,hit2_7) ||checkHits(hit1_0,hit2_8) ||checkHits(hit1_0,hit2_9);
//        //short matched_1 = checkHits(hit1_1,hit2_0) ||checkHits(hit1_1,hit2_1) ||checkHits(hit1_1,hit2_2) ||checkHits(hit1_1,hit2_3) ||checkHits(hit1_1,hit2_4) ||checkHits(hit1_1,hit2_5) ||checkHits(hit1_1,hit2_6) ||checkHits(hit1_1,hit2_7) ||checkHits(hit1_1,hit2_8) ||checkHits(hit1_1,hit2_9);
//        //short matched_2 = checkHits(hit1_2,hit2_0) ||checkHits(hit1_2,hit2_1) ||checkHits(hit1_2,hit2_2) ||checkHits(hit1_2,hit2_3) ||checkHits(hit1_2,hit2_4) ||checkHits(hit1_2,hit2_5) ||checkHits(hit1_2,hit2_6) ||checkHits(hit1_2,hit2_7) ||checkHits(hit1_2,hit2_8) ||checkHits(hit1_2,hit2_9);
//        //short matched_3 = checkHits(hit1_3,hit2_0) ||checkHits(hit1_3,hit2_1) ||checkHits(hit1_3,hit2_2) ||checkHits(hit1_3,hit2_3) ||checkHits(hit1_3,hit2_4) ||checkHits(hit1_3,hit2_5) ||checkHits(hit1_3,hit2_6) ||checkHits(hit1_3,hit2_7) ||checkHits(hit1_3,hit2_8) ||checkHits(hit1_3,hit2_9);
//        //short matched_4 = checkHits(hit1_4,hit2_0) ||checkHits(hit1_4,hit2_1) ||checkHits(hit1_4,hit2_2) ||checkHits(hit1_4,hit2_3) ||checkHits(hit1_4,hit2_4) ||checkHits(hit1_4,hit2_5) ||checkHits(hit1_4,hit2_6) ||checkHits(hit1_4,hit2_7) ||checkHits(hit1_4,hit2_8) ||checkHits(hit1_4,hit2_9);
//        //short matched_5 = checkHits(hit1_5,hit2_0) ||checkHits(hit1_5,hit2_1) ||checkHits(hit1_5,hit2_2) ||checkHits(hit1_5,hit2_3) ||checkHits(hit1_5,hit2_4) ||checkHits(hit1_5,hit2_5) ||checkHits(hit1_5,hit2_6) ||checkHits(hit1_5,hit2_7) ||checkHits(hit1_5,hit2_8) ||checkHits(hit1_5,hit2_9);
//        //short matched_6 = checkHits(hit1_6,hit2_0) ||checkHits(hit1_6,hit2_1) ||checkHits(hit1_6,hit2_2) ||checkHits(hit1_6,hit2_3) ||checkHits(hit1_6,hit2_4) ||checkHits(hit1_6,hit2_5) ||checkHits(hit1_6,hit2_6) ||checkHits(hit1_6,hit2_7) ||checkHits(hit1_6,hit2_8) ||checkHits(hit1_6,hit2_9);
//        //short matched_7 = checkHits(hit1_7,hit2_0) ||checkHits(hit1_7,hit2_1) ||checkHits(hit1_7,hit2_2) ||checkHits(hit1_7,hit2_3) ||checkHits(hit1_7,hit2_4) ||checkHits(hit1_7,hit2_5) ||checkHits(hit1_7,hit2_6) ||checkHits(hit1_7,hit2_7) ||checkHits(hit1_7,hit2_8) ||checkHits(hit1_7,hit2_9);
//        //short matched_8 = checkHits(hit1_8,hit2_0) ||checkHits(hit1_8,hit2_1) ||checkHits(hit1_8,hit2_2) ||checkHits(hit1_8,hit2_3) ||checkHits(hit1_8,hit2_4) ||checkHits(hit1_8,hit2_5) ||checkHits(hit1_8,hit2_6) ||checkHits(hit1_8,hit2_7) ||checkHits(hit1_8,hit2_8) ||checkHits(hit1_8,hit2_9);
//        //short matched_9 = checkHits(hit1_9,hit2_0) ||checkHits(hit1_9,hit2_1) ||checkHits(hit1_9,hit2_2) ||checkHits(hit1_9,hit2_3) ||checkHits(hit1_9,hit2_4) ||checkHits(hit1_9,hit2_5) ||checkHits(hit1_9,hit2_6) ||checkHits(hit1_9,hit2_7) ||checkHits(hit1_9,hit2_8) ||checkHits(hit1_9,hit2_9);
//        short matched_0 = checkHits(hit1_0,hit2_0) ||checkHits(hit1_0,hit2_2); 
//        short matched_1 = checkHits(hit1_1,hit2_1) ||checkHits(hit1_1,hit2_3);
//        short matched_2 = checkHits(hit1_2,hit2_0) ||checkHits(hit1_2,hit2_2) ||checkHits(hit1_2,hit2_4);
//        short matched_3 = checkHits(hit1_3,hit2_1) ||checkHits(hit1_3,hit2_3) ||checkHits(hit1_3,hit2_5);
//        short matched_4 = checkHits(hit1_4,hit2_2) ||checkHits(hit1_4,hit2_4) ||checkHits(hit1_4,hit2_6);
//        short matched_5 = checkHits(hit1_5,hit2_3) ||checkHits(hit1_5,hit2_5) ||checkHits(hit1_5,hit2_7);
//        short matched_6 = checkHits(hit1_6,hit2_4) ||checkHits(hit1_6,hit2_6) ||checkHits(hit1_6,hit2_8);
//        short matched_7 = checkHits(hit1_7,hit2_5) ||checkHits(hit1_7,hit2_7) ||checkHits(hit1_7,hit2_9);
//        short matched_8 = checkHits(hit1_8,hit2_6) ||checkHits(hit1_8,hit2_8);
//        short matched_9 = checkHits(hit1_9,hit2_7) ||checkHits(hit1_9,hit2_9);
//
//        if(matched_0+matched_1+matched_2+matched_3+matched_4+matched_5+matched_6+matched_7+matched_8+matched_9<8){continue;}
//        return;
//      }}
        short layer2_adjustment;
        if(modulesInGPU.layers[lowerModule1] == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
        else if( modulesInGPU.layers[lowerModule1] == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
        else{return;} // ignore anything else
       unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModuleArray1], 1);
       if(quintupletModuleIndex >= N_MAX_QUINTUPLETS_PER_MODULE)
       {
#ifdef Warnings
           if(quintupletModuleIndex ==  N_MAX_QUINTUPLETS_PER_MODULE)
               printf("Quintuplet excess alert! Module index = %d\n", lowerModuleArray1);
#endif
       }
       else
       {
           //this if statement should never get executed!
           if(modulesInGPU.quintupletModuleIndices[lowerModuleArray1] == -1)
           {
                printf("Quintuplets : no memory for module at module index = %d\n", lowerModuleArray1);
           }
           else
           {
                unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[lowerModuleArray1] +  quintupletModuleIndex;
#ifdef CUT_VALUE_DEBUG
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, quintupletIndex);
#else
        float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]]; 
        float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]]; 
        //float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]]]]; // inner triplet inner segment inner md inner hit
        //float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]+1]+1]]; // outer triplet outersegment outer md outer hit
        float pt = 0;  
        //printf("T5 %e %e %e\n",pt,eta,phi);
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, outerRadius, quintupletIndex,0,pt,eta,phi);
#endif
            }
        }
    }
}

#else
__global__ void createQuintupletsFromInnerInnerLowerModule(SDL::modules& modulesInGPU, SDL::hits& hitsInGPU, SDL::miniDoublets& mdsInGPU, SDL::segments& segmentsInGPU, SDL::triplets& tripletsInGPU, SDL::quintuplets& quintupletsInGPU, unsigned int lowerModuleArray1, unsigned int nInnerTriplets)
{
   int innerTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
   int outerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

   if(innerTripletArrayIndex >= nInnerTriplets) return;

   unsigned int innerTripletIndex = lowerModuleArray1 * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
   unsigned int lowerModule1 = modulesInGPU.lowerModuleIndices[lowerModuleArray1];
   //these are actual module indices!!! not lower module indices
   unsigned int lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
   unsigned int lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
   unsigned int lowerModuleArray3 = modulesInGPU.reverseLookupLowerModuleIndices[lowerModule3];

   unsigned int nOuterTriplets = min(tripletsInGPU.nTriplets[lowerModuleArray3], N_MAX_TRIPLETS_PER_MODULE);

   if(outerTripletArrayIndex >= nOuterTriplets) return;

   unsigned int outerTripletIndex = lowerModuleArray3 * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
    //these are actual module indices!!
    unsigned int lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
    unsigned int lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

    float innerRadius, innerRadiusMin, innerRadiusMin2S, innerRadiusMax, innerRadiusMax2S, outerRadius, outerRadiusMin, outerRadiusMin2S, outerRadiusMax, outerRadiusMax2S, bridgeRadius, bridgeRadiusMin, bridgeRadiusMin2S, bridgeRadiusMax, bridgeRadiusMax2S; //required for making distributions
    bool success = runQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S,
            outerRadiusMax2S);

   if(success)
   {
       unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModuleArray1], 1);
       if(quintupletModuleIndex >= N_MAX_QUINTUPLETS_PER_MODULE)
       {
#ifdef Warnings
           if(quintupletModuleIndex ==  N_MAX_QUINTUPLETS_PER_MODULE)
               printf("Quintuplet excess alert! Module index = %d\n", lowerModuleArray1);
#endif
       }
       else
       {
           if(modulesInGPU.quintupletModuleIndices[lowerModuleArray1] == -1)
           {
                printf("Quintuplets : no memory for module at module index = %d\n", lowerModuleArray1);
           }
           else
           {
                unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[lowerModuleArray1] +  quintupletModuleIndex;
#ifdef CUT_VALUE_DEBUG
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, quintupletIndex);
#else
        float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex]]]]; // inner triplet inner segment inner md inner hit
        float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex+1]+1]+1]]; // outer triplet outersegment outer md outer hit
        float pt = 0;
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, outerRadius, quintupletIndex,0,pt,eta,phi);
#endif

            }
        }
    }
}

__global__ void createQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU)
{
    int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; //inner triplet inner segment inner MD

    //no quintuplets can be formed for these folks - no need to run inner kernels for them!

    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules or modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1) return;

    unsigned int nInnerTriplets = min(tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex], N_MAX_TRIPLETS_PER_MODULE);
    if(nInnerTriplets == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerTriplets % nThreads.x == 0 ? nInnerTriplets / nThreads.x : nInnerTriplets / nThreads.x + 1, N_MAX_TRIPLETS_PER_MODULE % nThreads.y == 0 ? N_MAX_TRIPLETS_PER_MODULE / nThreads.y : N_MAX_TRIPLETS_PER_MODULE / nThreads.y + 1);

    createQuintupletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, innerInnerInnerLowerModuleArrayIndex, nInnerTriplets);

}

#endif

unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;

}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;

}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTracklets()
{
#ifdef Explicit_Tracklet
    unsigned int nLowerModules;// = *(SDL::modulesInGPU->nLowerModules);
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nTrackletsInPixelModule;
    hipMemcpy(&nTrackletsInPixelModule,pixelTrackletsInGPU->nPixelTracklets,sizeof(unsigned int),hipMemcpyDeviceToHost);
    return nTrackletsInPixelModule;
#else
    return *(pixelTrackletsInGPU->nPixelTracklets);
#endif

}

unsigned int SDL::Event::getNumberOfTracklets()
{
    unsigned int tracklets = 0;
    for(auto &it:n_tracklets_by_layer_barrel_)
    {
        tracklets += it;
    }
    for(auto &it:n_tracklets_by_layer_endcap_)
    {
        tracklets += it;
    }

    return tracklets;

}

unsigned int SDL::Event::getNumberOfTrackletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_tracklets_by_layer_barrel_[layer];
    else
        return n_tracklets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerBarrel(unsigned int layer)
{
    return n_tracklets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerEndcap(unsigned int layer)
{
    return n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;

}


unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTriplets()
{
#ifdef Explicit_PT3
    unsigned int nPixelTriplets;
    hipMemcpy(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
    return nPixelTriplets;
#else
    return *(pixelTripletsInGPU->nPixelTriplets); 
#endif
}

unsigned int SDL::Event::getNumberOfQuintuplets()
{
    unsigned int quintuplets = 0;
    for(auto &it:n_quintuplets_by_layer_barrel_)
    {
        quintuplets += it;
    }
    for(auto &it:n_quintuplets_by_layer_endcap_)
    {
        quintuplets += it;
    }

    return quintuplets;

}

unsigned int SDL::Event::getNumberOfQuintupletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_quintuplets_by_layer_barrel_[layer];
    else
        return n_quintuplets_by_layer_barrel_[layer] + n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerBarrel(unsigned int layer)
{
    return n_quintuplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerEndcap(unsigned int layer)
{
    return n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidates()
{
    unsigned int trackCandidates = 0;
    for(auto &it:n_trackCandidates_by_layer_barrel_)
    {
        trackCandidates += it;
    }
    for(auto &it:n_trackCandidates_by_layer_endcap_)
    {
        trackCandidates += it;
    }

    //hack - add pixel track candidate multiplicity
    trackCandidates += getNumberOfPixelTrackCandidates();

    return trackCandidates;

}

unsigned int SDL::Event::getNumberOfPixelTrackCandidates()
{
#ifdef Explicit_Track
    unsigned int nLowerModules;// = *(SDL::modulesInGPU->nLowerModules);
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nTrackCandidatesInPixelModule;
    hipMemcpy(&nTrackCandidatesInPixelModule,&trackCandidatesInGPU->nTrackCandidates[nLowerModules],sizeof(unsigned int),hipMemcpyDeviceToHost);
    return nTrackCandidatesInPixelModule;
#else
    return trackCandidatesInGPU->nTrackCandidates[*(modulesInGPU->nLowerModules)];
#endif

}
unsigned int SDL::Event::getNumberOfTrackCandidatesByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_trackCandidates_by_layer_barrel_[layer];
    else
        return n_trackCandidates_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerBarrel(unsigned int layer)
{
    return n_trackCandidates_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerEndcap(unsigned int layer)
{
    return n_trackCandidates_by_layer_endcap_[layer];
}

#ifdef Explicit_Hit
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpy(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost);
        *(hitsInCPU->nHits) = nHits;
        hitsInCPU->idxs = new unsigned int[nHits];
        hitsInCPU->xs = new float[nHits];
        hitsInCPU->ys = new float[nHits];
        hitsInCPU->zs = new float[nHits];
        hitsInCPU->moduleIndices = new unsigned int[nHits];
        hipMemcpy(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->xs, hitsInGPU->xs, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->ys, hitsInGPU->ys, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->zs, hitsInGPU->zs, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->moduleIndices, hitsInGPU->moduleIndices, sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost);
    }
    return hitsInCPU;
}
#else
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    return hitsInGPU;
}
#endif


#ifdef Explicit_MD
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        unsigned int nMemoryLocations = (N_MAX_MD_PER_MODULES * (nModules - 1) + N_MAX_PIXEL_MD_PER_MODULES);
        mdsInCPU->hitIndices = new unsigned int[2 * nMemoryLocations];
        mdsInCPU->nMDs = new unsigned int[nModules];
        hipMemcpy(mdsInCPU->hitIndices, mdsInGPU->hitIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(mdsInCPU->nMDs, mdsInGPU->nMDs, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return mdsInCPU;
}
#else
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}
#endif


#ifdef Explicit_Seg
SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        unsigned int nMemoryLocations = (N_MAX_SEGMENTS_PER_MODULE) * (nModules - 1) + N_MAX_PIXEL_SEGMENTS_PER_MODULE;
        segmentsInCPU->mdIndices = new unsigned int[2 * nMemoryLocations];
        segmentsInCPU->nSegments = new unsigned int[nModules];
        segmentsInCPU->innerMiniDoubletAnchorHitIndices = new unsigned int[nMemoryLocations];
        segmentsInCPU->outerMiniDoubletAnchorHitIndices = new unsigned int[nMemoryLocations];
        segmentsInCPU->ptIn = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->eta = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->phi = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        hipMemcpy(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->nSegments, segmentsInGPU->nSegments, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->innerMiniDoubletAnchorHitIndices, segmentsInGPU->innerMiniDoubletAnchorHitIndices, nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->outerMiniDoubletAnchorHitIndices, segmentsInGPU->outerMiniDoubletAnchorHitIndices, nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->ptIn, segmentsInGPU->ptIn, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->eta, segmentsInGPU->eta, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->phi, segmentsInGPU->phi, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);


    }
    return segmentsInCPU;
}
#else
SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}
#endif

#ifdef Explicit_Tracklet
SDL::tracklets* SDL::Event::getTracklets()
{
#ifdef FINAL_T3T4
    if(trackletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        trackletsInCPU = new SDL::tracklets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACKLETS_PER_MODULE) * nLowerModules;
        trackletsInCPU->segmentIndices = new unsigned int[2 * nMemoryLocations];
        trackletsInCPU->nTracklets = new unsigned int[nLowerModules];
        trackletsInCPU->betaIn = new float[nMemoryLocations];
        trackletsInCPU->betaOut = new float[nMemoryLocations];
        trackletsInCPU->pt_beta = new float[nMemoryLocations];
        hipMemcpy(trackletsInCPU->segmentIndices, trackletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->betaIn, trackletsInGPU->betaIn, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->betaOut, trackletsInGPU->betaOut, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->pt_beta, trackletsInGPU->pt_beta, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->nTracklets, trackletsInGPU->nTracklets, (nLowerModules)* sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
#endif
    return trackletsInCPU;
}

SDL::pixelTracklets* SDL::Event::getPixelTracklets()
{
    if(pixelTrackletsInCPU == nullptr)
    {
        pixelTrackletsInCPU = new SDL::pixelTracklets;
        pixelTrackletsInCPU->segmentIndices = new unsigned int[2 * N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->nPixelTracklets = new unsigned int;
        pixelTrackletsInCPU->betaIn = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->betaOut = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->pt_beta = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];

        hipMemcpy(pixelTrackletsInCPU->segmentIndices, pixelTrackletsInGPU->segmentIndices, 2 * N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->nPixelTracklets, pixelTrackletsInGPU->nPixelTracklets, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->betaIn, pixelTrackletsInGPU->betaIn, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->betaOut, pixelTrackletsInGPU->betaOut, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->pt_beta, pixelTrackletsInGPU->pt_beta, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
    }
    return pixelTrackletsInCPU;
}

#else
SDL::tracklets* SDL::Event::getTracklets()
{
    return trackletsInGPU;
}

SDL::pixelTracklets* SDL::Event::getPixelTracklets()
{
    return pixelTrackletsInGPU;
}
#endif

#ifdef Explicit_Trips
SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        tripletsInCPU = new SDL::triplets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRIPLETS_PER_MODULE) * (nLowerModules);
        tripletsInCPU->segmentIndices = new unsigned[2 * nMemoryLocations];
        tripletsInCPU->nTriplets = new unsigned int[nLowerModules];
        tripletsInCPU->betaIn = new float[nMemoryLocations];
        tripletsInCPU->betaOut = new float[nMemoryLocations];
        tripletsInCPU->pt_beta = new float[nMemoryLocations];
        hipMemcpy(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->betaIn, tripletsInGPU->betaIn, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->betaOut, tripletsInGPU->betaOut, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->pt_beta, tripletsInGPU->pt_beta, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return tripletsInCPU;
}
#else
SDL::triplets* SDL::Event::getTriplets()
{
    return tripletsInGPU;
}
#endif

#ifdef Explicit_T5
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    if(quintupletsInCPU == nullptr)
    {
        quintupletsInCPU = new SDL::quintuplets;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nEligibleT5Modules;
        hipMemcpy(&nEligibleT5Modules, modulesInGPU->nEligibleT5Modules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = nEligibleT5Modules * N_MAX_QUINTUPLETS_PER_MODULE;

        quintupletsInCPU->nQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->tripletIndices = new unsigned int[2 * nMemoryLocations];
        quintupletsInCPU->lowerModuleIndices = new unsigned int[5 * nMemoryLocations];
        quintupletsInCPU->innerRadius = new float[nMemoryLocations];
        quintupletsInCPU->outerRadius = new float[nMemoryLocations];
        hipMemcpy(quintupletsInCPU->nQuintuplets, quintupletsInGPU->nQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->tripletIndices, quintupletsInGPU->tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->lowerModuleIndices, quintupletsInGPU->lowerModuleIndices, 5 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->innerRadius, quintupletsInGPU->innerRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->outerRadius, quintupletsInGPU->outerRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
    }

    return quintupletsInCPU;
}
#else
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    return quintupletsInGPU;
}
#endif

#ifdef Explicit_PT3
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    if(pixelTripletsInCPU == nullptr)
    {
        pixelTripletsInCPU = new SDL::pixelTriplets;
        
        pixelTripletsInCPU->nPixelTriplets = new unsigned int;
        hipMemcpy(pixelTripletsInCPU->nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nPixelTriplets = *(pixelTripletsInCPU->nPixelTriplets);
        pixelTripletsInCPU->tripletIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelSegmentIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelRadius = new float[nPixelTriplets];
        pixelTripletsInCPU->pixelRadiusError = new float[nPixelTriplets];
        pixelTripletsInCPU->tripletRadius = new float[nPixelTriplets];

        hipMemcpy(pixelTripletsInCPU->tripletIndices, pixelTripletsInGPU->tripletIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->pixelSegmentIndices, pixelTripletsInGPU->pixelSegmentIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->pixelRadius, pixelTripletsInGPU->pixelRadius, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->tripletRadius, pixelTripletsInGPU->tripletRadius, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
    }
    return pixelTripletsInCPU;
}
#else
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    return pixelTripletsInGPU;
}
#endif


#ifdef Explicit_Track
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nEligibleModules;
        hipMemcpy(&nEligibleModules, modulesInGPU->nEligibleModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACK_CANDIDATES_PER_MODULE) * (nEligibleModules -1) + (N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE);

        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nMemoryLocations];
        trackCandidatesInCPU->trackCandidateType = new short[nMemoryLocations];
        trackCandidatesInCPU->nTrackCandidates = new unsigned int[nLowerModules+1];
        hipMemcpy(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nMemoryLocations * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, (nLowerModules + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return trackCandidatesInCPU;
}
#else
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    return trackCandidatesInGPU;
}
#endif
#ifdef Explicit_Module
SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    modulesInCPUFull->detIds = new unsigned int[nModules];
    modulesInCPUFull->moduleMap = new unsigned int[40*nModules];
    modulesInCPUFull->nConnectedModules = new unsigned int[nModules];
    modulesInCPUFull->drdzs = new float[nModules];
    modulesInCPUFull->slopes = new float[nModules];
    modulesInCPUFull->nModules = new unsigned int[1];
    modulesInCPUFull->nLowerModules = new unsigned int[1];
    modulesInCPUFull->layers = new short[nModules];
    modulesInCPUFull->rings = new short[nModules];
    modulesInCPUFull->modules = new short[nModules];
    modulesInCPUFull->rods = new short[nModules];
    modulesInCPUFull->subdets = new short[nModules];
    modulesInCPUFull->sides = new short[nModules];
    modulesInCPUFull->isInverted = new bool[nModules];
    modulesInCPUFull->isLower = new bool[nModules];

    modulesInCPUFull->hitRanges = new int[2*nModules];
    modulesInCPUFull->mdRanges = new int[2*nModules];
    modulesInCPUFull->segmentRanges = new int[2*nModules];
    modulesInCPUFull->trackletRanges = new int[2*nModules];
    modulesInCPUFull->tripletRanges = new int[2*nModules];
    modulesInCPUFull->trackCandidateRanges = new int[2*nModules];

    modulesInCPUFull->moduleType = new ModuleType[nModules];
    modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];

    modulesInCPUFull->lowerModuleIndices = new unsigned int[nLowerModules+1];
    modulesInCPUFull->reverseLookupLowerModuleIndices = new int[nModules];
    modulesInCPUFull->trackCandidateModuleIndices = new int[nLowerModules+1];
    modulesInCPUFull->quintupletModuleIndices = new int[nLowerModules];

    hipMemcpy(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->mdRanges, modulesInGPU->mdRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->segmentRanges, modulesInGPU->segmentRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackletRanges, modulesInGPU->trackletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->tripletRanges, modulesInGPU->tripletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateRanges, modulesInGPU->trackCandidateRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->reverseLookupLowerModuleIndices, modulesInGPU->reverseLookupLowerModuleIndices, nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost);
    }
    return modulesInCPUFull;
}
SDL::modules* SDL::Event::getModules()
{
    //if(modulesInCPU == nullptr)
    //{
        modulesInCPU = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        modulesInCPU->nLowerModules = new unsigned int[1];
        modulesInCPU->nModules = new unsigned int[1];
        modulesInCPU->lowerModuleIndices = new unsigned int[nLowerModules+1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->hitRanges = new int[2*nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->trackCandidateModuleIndices = new int[nLowerModules+1];
        modulesInCPU->quintupletModuleIndices = new int[nLowerModules];
        modulesInCPU->layers = new short[nModules];
        modulesInCPU->subdets = new short[nModules];
        modulesInCPU->rings = new short[nModules];


        hipMemcpy(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->layers, modulesInGPU->layers, nModules * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->subdets, modulesInGPU->subdets, nModules * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->rings, modulesInGPU->rings, nModules * sizeof(short), hipMemcpyDeviceToHost);
    //}
    return modulesInCPU;
}
#else
SDL::modules* SDL::Event::getModules()
{
    return modulesInGPU;
}
SDL::modules* SDL::Event::getFullModules()
{
    return modulesInGPU;
}
#endif
